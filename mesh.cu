#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

#include <hip/hip_complex.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h>
#include<ctime>
#include "mesh.h"
#include "numerical.h"

//cartCoord class functions
__host__ __device__ cartCoord::cartCoord(const cartCoord &rhs) {
    for(int i=0;i<3;i++) {
        coords[i] = rhs.coords[i];
    }
}

__host__ __device__ void cartCoord::set(const float x, const float y, const float z) {
    coords[0] = x;
    coords[1] = y;
    coords[2] = z;
}

ostream& operator<<(ostream &out,const cartCoord &rhs) {
    out << "(" << rhs.coords[0] << "," << rhs.coords[1] << "," << rhs.coords[2] 
            << ")";
    return out;
}

__host__ __device__ cartCoord& cartCoord::operator=(const cartCoord &rhs) {
    coords[0] = rhs.coords[0];
    coords[1] = rhs.coords[1];
    coords[2] = rhs.coords[2];
    return *this;
}

__host__ __device__ cartCoord cartCoord::operator+(const cartCoord &rhs) const {
    cartCoord temp;
    temp.coords[0] = coords[0] + rhs.coords[0];
    temp.coords[1] = coords[1] + rhs.coords[1];
    temp.coords[2] = coords[2] + rhs.coords[2];
    return temp;
}

__host__ __device__ cartCoord cartCoord::operator-(const cartCoord &rhs) const {
    cartCoord temp;
    temp.coords[0] = coords[0] - rhs.coords[0];
    temp.coords[1] = coords[1] - rhs.coords[1];
    temp.coords[2] = coords[2] - rhs.coords[2];
    return temp;
}

__host__ __device__ cartCoord cartCoord::operator-() const {
    return cartCoord(-coords[0],-coords[1],-coords[2]);
}

__host__ __device__ cartCoord cartCoord::operator*(const cartCoord &rhs) const {
    cartCoord prod;
    prod.coords[0] = coords[1]*rhs.coords[2]-coords[2]*rhs.coords[1];
    prod.coords[1] = coords[2]*rhs.coords[0]-coords[0]*rhs.coords[2];
    prod.coords[2] = coords[0]*rhs.coords[1]-coords[1]*rhs.coords[0];
    return prod;
}

__host__ __device__ float dotProd(const cartCoord &p1,const cartCoord &p2) {
    return p1.coords[0]*p2.coords[0]+p1.coords[1]*p2.coords[1]+p1.coords[2]*p2.coords[2];
}

__host__ __device__ cartCoord numDvd(const cartCoord &pnt, const float lambda) {
    if(lambda == 0) {
        printf("divisor cannot be 0.\n");
        return cartCoord(0,0,0);
    } else {
        return cartCoord(pnt.coords[0]/lambda,pnt.coords[1]/lambda,pnt.coords[2]/lambda);
    }
}

__host__ __device__ cartCoord numMul(const float lambda, const cartCoord &pnt) {
    return cartCoord(lambda*pnt.coords[0],lambda*pnt.coords[1],lambda*pnt.coords[2]);
}

__host__ __device__ float cartCoord::nrm2() const {
    return sqrtf(powf(coords[0],2)+powf(coords[1],2)+powf(coords[2],2));
}

__host__ __device__ float r(const cartCoord p1, const cartCoord p2) {
    cartCoord temp = p1-p2;
    return temp.nrm2();
}

__host__ __device__ float prpn2(const cartCoord n, const cartCoord p1, const cartCoord p2) {
    return ((p1.coords[0]-p2.coords[0])*n.coords[0]+(p1.coords[1]-p2.coords[1])*n.coords[1]
            +(p1.coords[2]-p2.coords[2])*n.coords[2])/r(p1,p2);
}

__host__ __device__ float prRpn2(const cartCoord n, const cartCoord p1, const cartCoord p2) {
    float temp1 = 1.0/powf(r(p1,p2),2), temp2 = prpn2(n,p1,p2);
    return -temp1*temp2;
}

__host__ __device__ hipFloatComplex green2(const float k, const cartCoord x, const cartCoord y) {
    cartCoord temp = x-y;
    float r = temp.nrm2();
    return green(k,r);
}

__host__ __device__ float PsiL2(const cartCoord p1, const cartCoord p2) {
    cartCoord temp = p1-p2;
    return PsiL(temp.nrm2());
}

__host__ __device__ float pPsiLpn2(const cartCoord n, const cartCoord p1, 
        const cartCoord p2) {
    return 1.0/(4*PI)*prRpn2(n,p1,p2);
}

__host__ __device__ hipFloatComplex pGpn2(const float k, const cartCoord n, 
        const cartCoord p1, const cartCoord p2) {
    hipFloatComplex temp1 = green2(k,p1,p2), temp2 = make_hipFloatComplex(-1.0/r(p1,p2),k);
    hipFloatComplex temp3 = hipCmulf(temp1,temp2);
    float temp4 = prpn2(n,p1,p2);
    return make_hipFloatComplex(temp4*hipCrealf(temp3),temp4*hipCimagf(temp3));
}

__device__ void g_l_nsgl(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *gCoeff1, 
        hipFloatComplex *gCoeff2, hipFloatComplex *gCoeff3) {
    *gCoeff1 = make_hipFloatComplex(0,0);
    *gCoeff2 = make_hipFloatComplex(0,0);
    *gCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4], 
            omega = k*speed;
    cartCoord y, crossProd;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*(1-theta);
            xi2 = rho*theta;
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = green2(k,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*density*omega;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *gCoeff1 = hipCaddf(*gCoeff1,make_hipFloatComplex(-temp[1]*hipCimagf(g),
                    temp[1]*hipCrealf(g)));
            *gCoeff2 = hipCaddf(*gCoeff2,make_hipFloatComplex(-temp[2]*hipCimagf(g),
                    temp[2]*hipCrealf(g)));
            *gCoeff3 = hipCaddf(*gCoeff3,make_hipFloatComplex(-temp[3]*hipCimagf(g),
                    temp[3]*hipCrealf(g)));
        }
    }
}

__device__ void h_l_nsgl(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *hCoeff1, 
        hipFloatComplex *hCoeff2, hipFloatComplex *hCoeff3) {
    *hCoeff1 = make_hipFloatComplex(0,0);
    *hCoeff2 = make_hipFloatComplex(0,0);
    *hCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4];
    cartCoord y, crossProd, normal;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*(1-theta);
            xi2 = rho*theta;
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = pGpn2(k,normal,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *hCoeff1 = hipCaddf(*hCoeff1,make_hipFloatComplex(temp[1]*hipCrealf(g),
                    temp[1]*hipCimagf(g)));
            *hCoeff2 = hipCaddf(*hCoeff2,make_hipFloatComplex(temp[2]*hipCrealf(g),
                    temp[2]*hipCimagf(g)));
            *hCoeff3 = hipCaddf(*hCoeff3,make_hipFloatComplex(temp[3]*hipCrealf(g),
                    temp[3]*hipCimagf(g)));
        }
    }
}

__device__ void g_l_sgl1(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *gCoeff1, 
        hipFloatComplex *gCoeff2, hipFloatComplex *gCoeff3) {
    *gCoeff1 = make_hipFloatComplex(0,0);
    *gCoeff2 = make_hipFloatComplex(0,0);
    *gCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4], 
            omega = k*speed;
    cartCoord y, crossProd;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = 1-rho;
            xi2 = rho*(1-theta);
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = green2(k,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*density*omega;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *gCoeff1 = hipCaddf(*gCoeff1,make_hipFloatComplex(-temp[1]*hipCimagf(g),
                    temp[1]*hipCrealf(g)));
            *gCoeff2 = hipCaddf(*gCoeff2,make_hipFloatComplex(-temp[2]*hipCimagf(g),
                    temp[2]*hipCrealf(g)));
            *gCoeff3 = hipCaddf(*gCoeff3,make_hipFloatComplex(-temp[3]*hipCimagf(g),
                    temp[3]*hipCrealf(g)));
        }
    }
}

__device__ void g_l_sgl2(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *gCoeff1, 
        hipFloatComplex *gCoeff2, hipFloatComplex *gCoeff3) {
    *gCoeff1 = make_hipFloatComplex(0,0);
    *gCoeff2 = make_hipFloatComplex(0,0);
    *gCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4], 
            omega = k*speed;
    cartCoord y, crossProd;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*theta;
            xi2 = 1-rho;
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = green2(k,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*density*omega;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *gCoeff1 = hipCaddf(*gCoeff1,make_hipFloatComplex(-temp[1]*hipCimagf(g),
                    temp[1]*hipCrealf(g)));
            *gCoeff2 = hipCaddf(*gCoeff2,make_hipFloatComplex(-temp[2]*hipCimagf(g),
                    temp[2]*hipCrealf(g)));
            *gCoeff3 = hipCaddf(*gCoeff3,make_hipFloatComplex(-temp[3]*hipCimagf(g),
                    temp[3]*hipCrealf(g)));
        }
    }
}

__device__ void g_l_sgl3(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *gCoeff1, 
        hipFloatComplex *gCoeff2, hipFloatComplex *gCoeff3) {
    *gCoeff1 = make_hipFloatComplex(0,0);
    *gCoeff2 = make_hipFloatComplex(0,0);
    *gCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4], 
            omega = k*speed;
    cartCoord y, crossProd;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*(1-theta);
            xi2 = rho*theta;
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = green2(k,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*density*omega;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *gCoeff1 = hipCaddf(*gCoeff1,make_hipFloatComplex(-temp[1]*hipCimagf(g),
                    temp[1]*hipCrealf(g)));
            *gCoeff2 = hipCaddf(*gCoeff2,make_hipFloatComplex(-temp[2]*hipCimagf(g),
                    temp[2]*hipCrealf(g)));
            *gCoeff3 = hipCaddf(*gCoeff3,make_hipFloatComplex(-temp[3]*hipCimagf(g),
                    temp[3]*hipCrealf(g)));
        }
    }
}

__device__ void h_l_sgl1(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *hCoeff1, 
        hipFloatComplex *hCoeff2, hipFloatComplex *hCoeff3) {
    *hCoeff1 = make_hipFloatComplex(0,0);
    *hCoeff2 = make_hipFloatComplex(0,0);
    *hCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4];
    cartCoord y, crossProd, normal;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = 1-rho;
            xi2 = rho*(1-theta);
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = pGpn2(k,normal,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *hCoeff1 = hipCaddf(*hCoeff1,make_hipFloatComplex(temp[1]*hipCrealf(g),
                    temp[1]*hipCimagf(g)));
            *hCoeff2 = hipCaddf(*hCoeff2,make_hipFloatComplex(temp[2]*hipCrealf(g),
                    temp[2]*hipCimagf(g)));
            *hCoeff3 = hipCaddf(*hCoeff3,make_hipFloatComplex(temp[3]*hipCrealf(g),
                    temp[3]*hipCimagf(g)));
        }
    }
}

__device__ void h_l_sgl2(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *hCoeff1, 
        hipFloatComplex *hCoeff2, hipFloatComplex *hCoeff3) {
    *hCoeff1 = make_hipFloatComplex(0,0);
    *hCoeff2 = make_hipFloatComplex(0,0);
    *hCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4];
    cartCoord y, crossProd, normal;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*theta;
            xi2 = 1-rho;
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = pGpn2(k,normal,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *hCoeff1 = hipCaddf(*hCoeff1,make_hipFloatComplex(temp[1]*hipCrealf(g),
                    temp[1]*hipCimagf(g)));
            *hCoeff2 = hipCaddf(*hCoeff2,make_hipFloatComplex(temp[2]*hipCrealf(g),
                    temp[2]*hipCimagf(g)));
            *hCoeff3 = hipCaddf(*hCoeff3,make_hipFloatComplex(temp[3]*hipCrealf(g),
                    temp[3]*hipCimagf(g)));
        }
    }
}

__device__ void h_l_sgl3(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, hipFloatComplex *hCoeff1, 
        hipFloatComplex *hCoeff2, hipFloatComplex *hCoeff3) {
    *hCoeff1 = make_hipFloatComplex(0,0);
    *hCoeff2 = make_hipFloatComplex(0,0);
    *hCoeff3 = make_hipFloatComplex(0,0);
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, N1, N2, N3, temp[4];
    cartCoord y, crossProd, normal;
    hipFloatComplex g;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*(1-theta);
            xi2 = rho*theta;
            N1 = N_1(cartCoord2D(xi1,xi2));
            N2 = N_2(cartCoord2D(xi1,xi2));
            N3 = N_3(cartCoord2D(xi1,xi2));
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            g = pGpn2(k,normal,x,y);
            temp[0] = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            temp[1] = temp[0]*N1;
            temp[2] = temp[0]*N2;
            temp[3] = temp[0]*N3;
            *hCoeff1 = hipCaddf(*hCoeff1,make_hipFloatComplex(temp[1]*hipCrealf(g),
                    temp[1]*hipCimagf(g)));
            *hCoeff2 = hipCaddf(*hCoeff2,make_hipFloatComplex(temp[2]*hipCrealf(g),
                    temp[2]*hipCimagf(g)));
            *hCoeff3 = hipCaddf(*hCoeff3,make_hipFloatComplex(temp[3]*hipCrealf(g),
                    temp[3]*hipCimagf(g)));
        }
    }
}

__device__ void c_l_nsgl(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, float *cCoeff) {
    *cCoeff = 0;
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, temp;
    cartCoord y, crossProd, normal;
    float psi;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*(1-theta);
            xi2 = rho*theta;
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            psi = pPsiLpn2(normal,x,y);
            temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            *cCoeff += temp*psi;
        }
    }
}

__device__ void c_l_sgl1(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, float *cCoeff) {
    *cCoeff = 0;
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, temp;
    cartCoord y, crossProd, normal;
    float psi;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = 1-rho;
            xi2 = rho*(1-theta);
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            psi = pPsiLpn2(normal,x,y);
            temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            *cCoeff += temp*psi;
        }
    }
}

__device__ void c_l_sgl2(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, float *cCoeff) {
    *cCoeff = 0;
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, temp;
    cartCoord y, crossProd, normal;
    float psi;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*theta;
            xi2 = 1-rho;
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            psi = pPsiLpn2(normal,x,y);
            temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            *cCoeff += temp*psi;
        }
    }
}

__device__ void c_l_sgl3(const float k, const cartCoord x, const cartCoord p1, 
        const cartCoord p2, const cartCoord p3, float *cCoeff) {
    *cCoeff = 0;
    float eta1, eta2, xi1, xi2, rho, theta, vertCrossProd, temp;
    cartCoord y, crossProd, normal;
    float psi;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    normal = crossProd.nrmlzd();
    int n, m;
    for(n=0;n<INTORDER;n++) {
        eta2 = INTPNTS[n];
        for(m=0;m<INTORDER;m++) {
            eta1 = INTPNTS[m];
            rho = 0.5+0.5*eta1;
            theta = 0.5+0.5*eta2;
            xi1 = rho*(1-theta);
            xi2 = rho*theta;
            y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
            psi = pPsiLpn2(normal,x,y);
            temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd;
            *cCoeff += temp*psi;
        }
    }
}

//singular not dealt with in this function
__global__ void elemLPnts_nsgl(const float k, const int l, const triElem *elems, const cartCoord *pnts, 
        const int numNods, const int numCHIEF, hipFloatComplex *A, const int lda, 
        hipFloatComplex *B, const int numSrcs, const int ldb) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numNods+numCHIEF && idx!=elems[l].nodes[0] && idx!=elems[l].nodes[1] && idx!=elems[l].nodes[2]) {
        int i, j;
        hipFloatComplex hCoeffs[3], gCoeffs[3], bc, pCoeffs[3];
        float cCoeff;
        h_l_nsgl(k,pnts[idx],pnts[elems[l].nodes[0]],pnts[elems[l].nodes[1]],pnts[elems[l].nodes[2]],
                hCoeffs,hCoeffs+1,hCoeffs+2);
        g_l_nsgl(k,pnts[idx],pnts[elems[l].nodes[0]],pnts[elems[l].nodes[1]],pnts[elems[l].nodes[2]],
                gCoeffs,gCoeffs+1,gCoeffs+2);
        c_l_nsgl(k,pnts[idx],pnts[elems[l].nodes[0]],pnts[elems[l].nodes[1]],pnts[elems[l].nodes[2]],
                &cCoeff); 
        
        //Update the A matrix
        bc = hipCdivf(elems[l].bc[0],elems[l].bc[1]);
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeffs[i],hipCmulf(bc,gCoeffs[i]));
        }
        for(i=0;i<3;i++) {
            A[IDXC0(idx,elems[l].nodes[i],lda)] = hipCaddf(A[IDXC0(idx,elems[l].nodes[i],lda)],pCoeffs[i]);
        }
        A[IDXC0(idx,idx,lda)] = hipCsubf(A[IDXC0(idx,idx,lda)],make_hipFloatComplex(cCoeff,0));
        
        //Update the B matrix
        bc = hipCdivf(elems[l].bc[2],elems[l].bc[1]);
        for(i=0;i<numSrcs;i++) {
            for(j=0;j<3;j++) {
                B[IDXC0(idx,i,ldb)] = hipCsubf(B[IDXC0(idx,i,ldb)],hipCmulf(bc,gCoeffs[i]));
            }
        }       
    }
}

__global__ void elemsPnts_sgl(const float k, const triElem *elems, const int numElems,
        const cartCoord *pnts, hipFloatComplex *hCoeffs_sgl1, hipFloatComplex *hCoeffs_sgl2, 
        hipFloatComplex *hCoeffs_sgl3, hipFloatComplex *gCoeffs_sgl1, hipFloatComplex *gCoeffs_sgl2, 
        hipFloatComplex *gCoeffs_sgl3, float *cCoeffs_sgl1, float *cCoeffs_sgl2, float *cCoeffs_sgl3) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElems) {
        triElem elem = elems[idx];
        h_l_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                hCoeffs_sgl1+3*idx,hCoeffs_sgl1+3*idx+1,hCoeffs_sgl1+3*idx+2);
        h_l_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                hCoeffs_sgl2+3*idx,hCoeffs_sgl2+3*idx+1,hCoeffs_sgl2+3*idx+2);
        h_l_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                hCoeffs_sgl3+3*idx,hCoeffs_sgl3+3*idx+1,hCoeffs_sgl3+3*idx+2);
        
        g_l_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                gCoeffs_sgl1+3*idx,gCoeffs_sgl1+3*idx+1,gCoeffs_sgl1+3*idx+2);
        g_l_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                gCoeffs_sgl2+3*idx,gCoeffs_sgl2+3*idx+1,gCoeffs_sgl2+3*idx+2);
        g_l_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                gCoeffs_sgl3+3*idx,gCoeffs_sgl3+3*idx+1,gCoeffs_sgl3+3*idx+2);
        
        c_l_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                cCoeffs_sgl1+idx);
        c_l_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                cCoeffs_sgl2+idx);
        c_l_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                cCoeffs_sgl3+idx);
    }
}

__global__ void updateSystem_sgl(const triElem *elems, const int numElems, hipFloatComplex *hCoeffs_sgl1, 
        hipFloatComplex *hCoeffs_sgl2, hipFloatComplex *hCoeffs_sgl3, hipFloatComplex *gCoeffs_sgl1, 
        hipFloatComplex *gCoeffs_sgl2, hipFloatComplex *gCoeffs_sgl3, hipFloatComplex *A, const int lda) {
    //Indices with the same row and column index has to be updated on the CPU!
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElems) {
        int i;
        triElem elem = elems[idx];
        hipFloatComplex pCoeffs[3];
        hipFloatComplex bc = hipCdivf(elem.bc[0],elem.bc[1]);
        
        //Deal with singularity 1
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeffs_sgl1[3*idx+i],hipCmulf(bc,gCoeffs_sgl1[3*idx+i]));
        }
        A[IDXC0(elem.nodes[0],elem.nodes[1],lda)] = hipCaddf(A[IDXC0(elem.nodes[0],elem.nodes[1],lda)],
                pCoeffs[1]);
        A[IDXC0(elem.nodes[0],elem.nodes[2],lda)] = hipCaddf(A[IDXC0(elem.nodes[0],elem.nodes[2],lda)],
                pCoeffs[2]);
        
        //Deal with singularity 2
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeffs_sgl2[3*idx+i],hipCmulf(bc,gCoeffs_sgl2[3*idx+i]));
        }
        A[IDXC0(elem.nodes[1],elem.nodes[0],lda)] = hipCaddf(A[IDXC0(elem.nodes[1],elem.nodes[0],lda)],
                pCoeffs[0]);
        A[IDXC0(elem.nodes[1],elem.nodes[2],lda)] = hipCaddf(A[IDXC0(elem.nodes[1],elem.nodes[2],lda)],
                pCoeffs[2]);
        
        //Deal with singularity 3
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeffs_sgl3[3*idx+i],hipCmulf(bc,gCoeffs_sgl3[3*idx+i]));
        }
        A[IDXC0(elem.nodes[2],elem.nodes[0],lda)] = hipCaddf(A[IDXC0(elem.nodes[2],elem.nodes[0],lda)],
                pCoeffs[0]);
        A[IDXC0(elem.nodes[2],elem.nodes[1],lda)] = hipCaddf(A[IDXC0(elem.nodes[2],elem.nodes[1],lda)],
                pCoeffs[1]);
    }
}

void updateSystemCPU(const triElem *elems, const int numElems, 
        hipFloatComplex *hCoeffs_sgl1, hipFloatComplex *hCoeffs_sgl2, hipFloatComplex *hCoeffs_sgl3, 
        hipFloatComplex *gCoeffs_sgl1, hipFloatComplex *gCoeffs_sgl2, hipFloatComplex *gCoeffs_sgl3, 
        float *cCoeffs_sgl1, float *cCoeffs_sgl2, float *cCoeffs_sgl3,
        hipFloatComplex *A, const int lda, hipFloatComplex *B, const int numSrcs, const int ldb) {
    int i, j, k;
    hipFloatComplex bc, pCoeff;
    for(i=0;i<numElems;i++) {
        bc = hipCdivf(elems[i].bc[0],elems[i].bc[1]);
        pCoeff = hipCsubf(hCoeffs_sgl1[3*i],hipCmulf(bc,gCoeffs_sgl1[3*i]));
        A[IDXC0(elems[i].nodes[0],elems[i].nodes[0],lda)] = hipCaddf(A[IDXC0(elems[i].nodes[0],elems[i].nodes[0],lda)],
            pCoeff);
        pCoeff = hipCsubf(hCoeffs_sgl2[3*i+1],hipCmulf(bc,gCoeffs_sgl2[3*i+1]));
        A[IDXC0(elems[i].nodes[1],elems[i].nodes[1],lda)] = hipCaddf(A[IDXC0(elems[i].nodes[1],elems[i].nodes[1],lda)],
            pCoeff);
        pCoeff = hipCsubf(hCoeffs_sgl3[3*i+2],hipCmulf(bc,gCoeffs_sgl3[3*i+2]));
        A[IDXC0(elems[i].nodes[2],elems[i].nodes[2],lda)] = hipCaddf(A[IDXC0(elems[i].nodes[2],elems[i].nodes[2],lda)],
            pCoeff);
        
        A[IDXC0(elems[i].nodes[0],elems[i].nodes[0],lda)] = hipCsubf(A[IDXC0(elems[i].nodes[0],elems[i].nodes[0],lda)],
                make_hipFloatComplex(cCoeffs_sgl1[i],0));
        A[IDXC0(elems[i].nodes[1],elems[i].nodes[1],lda)] = hipCsubf(A[IDXC0(elems[i].nodes[1],elems[i].nodes[1],lda)],
                make_hipFloatComplex(cCoeffs_sgl2[i],0));
        A[IDXC0(elems[i].nodes[2],elems[i].nodes[2],lda)] = hipCsubf(A[IDXC0(elems[i].nodes[2],elems[i].nodes[2],lda)],
                make_hipFloatComplex(cCoeffs_sgl3[i],0));
        
        
        bc = hipCdivf(elems[i].bc[2],elems[i].bc[1]);
        for(j=0;j<numSrcs;j++) {
            for(k=0;k<3;k++) {
                B[IDXC0(elems[i].nodes[0],j,ldb)] = hipCsubf(B[IDXC0(elems[i].nodes[0],j,ldb)],
                    hipCmulf(bc,gCoeffs_sgl1[3*i+k]));
                B[IDXC0(elems[i].nodes[1],j,ldb)] = hipCsubf(B[IDXC0(elems[i].nodes[1],j,ldb)],
                    hipCmulf(bc,gCoeffs_sgl2[3*i+k]));
                B[IDXC0(elems[i].nodes[2],j,ldb)] = hipCsubf(B[IDXC0(elems[i].nodes[2],j,ldb)],
                    hipCmulf(bc,gCoeffs_sgl3[3*i+k]));
            }
        }
    }
}

int genSystem(const float k, const triElem *elems, const int numElems, 
        const cartCoord *pnts, const int numNods, const int numCHIEF, 
        const cartCoord *srcs, const int numSrcs, hipFloatComplex *A, const int lda, 
        hipFloatComplex *B, const int ldb) {
    //Initialization of A
    int i, j, l;
    for(i=0;i<numNods+numCHIEF;i++) {
        for(j=0;j<numNods;j++) {
            if(i==j) {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(1,0);
            } else {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(0,0);
            }
        }
    }
    
    //Initialization of B
    for(i=0;i<numNods+numCHIEF;i++) {
        for(j=0;j<numSrcs;j++) {
            B[IDXC0(i,j,ldb)] = green2(k,srcs[j],pnts[i]);
        }
    }
    
    triElem *elems_d;
    CUDA_CALL(hipMalloc(&elems_d,numElems*sizeof(triElem)));
    CUDA_CALL(hipMemcpy(elems_d,elems,numElems*sizeof(triElem),hipMemcpyHostToDevice));
    
    cartCoord *pnts_d;
    CUDA_CALL(hipMalloc(&pnts_d,(numNods+numCHIEF)*sizeof(cartCoord)));
    CUDA_CALL(hipMemcpy(pnts_d,pnts,(numNods+numCHIEF)*sizeof(cartCoord),hipMemcpyHostToDevice));
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNods+numCHIEF)*numNods*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNods+numCHIEF)*numNods*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNods+numCHIEF)*numSrcs*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNods+numCHIEF)*numSrcs*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int numBlocks, width = 32;
    numBlocks = (numNods+numCHIEF+width-1)/width;
    
    for(l=0;l<numElems;l++) {
        elemLPnts_nsgl<<<numBlocks,width>>>(k,l,elems_d,pnts_d,numNods,numCHIEF,A_d,lda,B_d,numSrcs,ldb);
    }
    
    //Update singular
    hipFloatComplex *hCoeffs_sgl1, *hCoeffs_sgl2, *hCoeffs_sgl3, *gCoeffs_sgl1, 
            *gCoeffs_sgl2, *gCoeffs_sgl3, *hCoeffs_sgl1_d, *hCoeffs_sgl2_d, 
            *hCoeffs_sgl3_d, *gCoeffs_sgl1_d, *gCoeffs_sgl2_d, *gCoeffs_sgl3_d;
    float *cCoeffs_sgl1, *cCoeffs_sgl2, *cCoeffs_sgl3, 
            *cCoeffs_sgl1_d, *cCoeffs_sgl2_d, *cCoeffs_sgl3_d;
    
    hCoeffs_sgl1 = new hipFloatComplex[3*numElems];
    hCoeffs_sgl2 = new hipFloatComplex[3*numElems];
    hCoeffs_sgl3 = new hipFloatComplex[3*numElems];
    gCoeffs_sgl1 = new hipFloatComplex[3*numElems];
    gCoeffs_sgl2 = new hipFloatComplex[3*numElems];
    gCoeffs_sgl3 = new hipFloatComplex[3*numElems];
    
    cCoeffs_sgl1 = new float[numElems];
    cCoeffs_sgl2 = new float[numElems];
    cCoeffs_sgl3 = new float[numElems];
    
    CUDA_CALL(hipMalloc(&hCoeffs_sgl1_d,3*numElems*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMalloc(&hCoeffs_sgl2_d,3*numElems*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMalloc(&hCoeffs_sgl3_d,3*numElems*sizeof(hipFloatComplex)));
    
    CUDA_CALL(hipMalloc(&gCoeffs_sgl1_d,3*numElems*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMalloc(&gCoeffs_sgl2_d,3*numElems*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMalloc(&gCoeffs_sgl3_d,3*numElems*sizeof(hipFloatComplex)));
    
    CUDA_CALL(hipMalloc(&cCoeffs_sgl1_d,numElems*sizeof(float)));
    CUDA_CALL(hipMalloc(&cCoeffs_sgl2_d,numElems*sizeof(float)));
    CUDA_CALL(hipMalloc(&cCoeffs_sgl3_d,numElems*sizeof(float)));
    
    numBlocks = (numElems+width-1)/width;
    elemsPnts_sgl<<<numBlocks,width>>>(k,elems_d,numElems,pnts_d,hCoeffs_sgl1_d,hCoeffs_sgl2_d,hCoeffs_sgl3_d,
            gCoeffs_sgl1_d,gCoeffs_sgl2_d,gCoeffs_sgl3_d,cCoeffs_sgl1_d,cCoeffs_sgl2_d,
            cCoeffs_sgl3_d);
    
    CUDA_CALL(hipMemcpy(hCoeffs_sgl1,hCoeffs_sgl1_d,3*numElems*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(hCoeffs_sgl2,hCoeffs_sgl2_d,3*numElems*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(hCoeffs_sgl3,hCoeffs_sgl3_d,3*numElems*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipMemcpy(gCoeffs_sgl1,gCoeffs_sgl1_d,3*numElems*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(gCoeffs_sgl2,gCoeffs_sgl2_d,3*numElems*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(gCoeffs_sgl3,gCoeffs_sgl3_d,3*numElems*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipMemcpy(cCoeffs_sgl1,cCoeffs_sgl1_d,numElems*sizeof(float),
            hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(cCoeffs_sgl2,cCoeffs_sgl2_d,numElems*sizeof(float),
            hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(cCoeffs_sgl3,cCoeffs_sgl3_d,numElems*sizeof(float),
            hipMemcpyDeviceToHost));
    
    updateSystem_sgl<<<numBlocks,width>>>(elems_d,numElems,hCoeffs_sgl1_d,hCoeffs_sgl2_d,hCoeffs_sgl3_d,
            gCoeffs_sgl1_d,gCoeffs_sgl2_d,gCoeffs_sgl3_d,A_d,lda);
    
    updateSystemCPU(elems,numElems,hCoeffs_sgl1,hCoeffs_sgl2,hCoeffs_sgl3,
            gCoeffs_sgl1,gCoeffs_sgl2,gCoeffs_sgl3,cCoeffs_sgl1,cCoeffs_sgl2,cCoeffs_sgl3,
            A,lda,B,numSrcs,ldb);
    
    return EXIT_SUCCESS;
}

__host__ __device__ float trnglArea(const cartCoord p1, const cartCoord p2) {
    cartCoord temp = p1*p2;
    return temp.nrm2()/2.0;
}

__host__ __device__ cartCoord cartCoord::nrmlzd() {
    if(nrm2() == 0) {
        return cartCoord(nanf(""),nanf(""),nanf(""));
    } else {
        float nrm = nrm2();
        return numDvd(*this,nrm);
    }
}

__host__ __device__ cartCoord rayPlaneInt(const cartCoord sp, const cartCoord dir,
    const cartCoord n, const cartCoord pnt) {
    cartCoord temp;
    if(dotProd(n,sp-pnt) == 0) {
        temp =  sp;
    } else {
        if(dotProd(n,dir)==0) {
            temp =  cartCoord(nanf(""),nanf(""),nanf(""));
        } else {
            float t = (dotProd(n,pnt)-dotProd(n,sp))/dotProd(n,dir);
            if(t>0) {
                temp = sp+numMul(t,dir);
            } else {
                temp = cartCoord(nanf(""),nanf(""),nanf(""));
            }
        }
    }
    //printf("(%f,%f,%f)\n",temp.coords[0],temp.coords[1],temp.coords[2]);
    return temp;
}

__host__ __device__ bool cartCoord::isLegal() const {
    if(coords[0]!=coords[0]||coords[1]!=coords[1]||coords[2]!=coords[2]) {
        return false;
    } else {
        return true;
    }
}

__host__ __device__ bool cartCoord::isEqual(const cartCoord p) const {
    if(abs(coords[0]-p.coords[0])<EPS && abs(coords[1]-p.coords[1])<EPS 
            && abs(coords[2]-p.coords[2])<EPS) {
        return true;
    } else {
        return false;
    }
}


__host__ __device__ bool cartCoord::isInsideTrngl(const cartCoord p1,
        const cartCoord p2, const cartCoord p3) const {
    if(!isLegal()) {
        return false;
    } else {
        cartCoord v12 = p2-p1, v23 = p3-p2, v31 = p1-p3;
        cartCoord v1p = *this-p1, v2p = *this-p2, v3p = *this-p3;
        cartCoord t1 = v12*v1p, t2 = v23*v2p, t3 = v31*v3p;
        cartCoord t1Nrm = t1.nrmlzd(), t2Nrm = t2.nrmlzd(), t3Nrm = t3.nrmlzd();
        if(t1Nrm.isEqual(t2Nrm) && t2Nrm.isEqual(t3Nrm)) {
            return true;
        } else {
            return false;
        }
    }
}

__host__ __device__ bool rayTrnglInt(const cartCoord sp, const cartCoord dir,
    const cartCoord p1, const cartCoord p2, const cartCoord p3) {
    cartCoord n = (p2-p1)*(p3-p1);
    cartCoord intPnt = rayPlaneInt(sp,dir,n,p1);
    return intPnt.isInsideTrngl(p1,p2,p3);
}

//triangular element class
__host__ __device__ triElem::triElem(const triElem &rhs) {
    for(int i=0;i<3;i++) {
        nodes[i] = rhs.nodes[i];
        bc[i] = rhs.bc[i];
    }
}

__host__ __device__ triElem& triElem::operator=(const triElem &rhs) {
    for(int i=0;i<3;i++) {
        nodes[i] = rhs.nodes[i];
        bc[i] = rhs.bc[i];
    }
    return *this;
}

ostream& operator<<(ostream &out, const triElem &rhs) {
    out << "nodes indices: " << rhs.nodes[0] << ", " << rhs.nodes[1] << ", " 
            << rhs.nodes[2] << " boundary condition: " << "A = " << rhs.bc[0] 
            << ", B = " << rhs.bc[1] << ", C = " << rhs.bc[2];
    return out;
}

//class mesh
int mesh::readObj(const char *file) {
    int temp[3];
    vector<cartCoord> pntVec; 
    vector<triElem> elemVec;
    cartCoord pnt;
    triElem elem;
    FILE *fp = fopen(file,"r");
    if (fp == NULL) {
        printf("Failed to open file.\n");
        return EXIT_FAILURE;
    }
    int i = 0;
    char line[50];
    char type[5];
    while (fgets(line,49,fp) != NULL) {
        if (line[0] == 'v') {
            sscanf(line,"%s %f %f %f",type,&pnt.coords[0],&pnt.coords[1],&pnt.coords[2]);
            pntVec.push_back(pnt);
        }

        if (line[0] == 'f') {
            sscanf(line,"%s %d %d %d",type,&temp[0],&temp[1],&temp[2]);
            elem.nodes[0] = temp[0]-1;
            elem.nodes[1] = temp[1]-1;
            elem.nodes[2] = temp[2]-1;
            elem.bc[0] = make_hipFloatComplex(0,0); // ca=0
            elem.bc[1] = make_hipFloatComplex(1,0); // cb=1
            elem.bc[2] = make_hipFloatComplex(0,0); // cc=0
            elemVec.push_back(elem);
        }
    }
    if(pnts != NULL) {
        delete[] pnts;
    }
    pnts = new cartCoord[pntVec.size()];
    for(i=0;i<pntVec.size();i++) {
        pnts[i] = pntVec[i];
    }
    numPnts = pntVec.size();
    
    if(elems != NULL) {
        delete[] elems;
    }
    elems = new triElem[elemVec.size()];
    for(i=0;i<elemVec.size();i++) {
        elems[i] = elemVec[i];
    }
    numElems = elemVec.size();
    fclose(fp);
    return EXIT_SUCCESS;
}

mesh::mesh(const mesh &rhs) {
    if(rhs.numPnts > 0) {
        if(pnts != NULL) {
            delete[] pnts;
        }
        numPnts = rhs.numPnts;
        pnts = new cartCoord[numPnts];
        for(int i=0;i<numPnts;i++) {
            pnts[i] = rhs.pnts[i];
        }
    } else {
        if(pnts != NULL) {
            delete[] pnts;
        }
    }
    
    if(rhs.numElems > 0) {
        if(elems != NULL) {
            delete[] elems;
        }
        numElems = rhs.numElems;
        elems = new triElem[numElems];
        for(int i=0;i<numPnts;i++) {
            elems[i] = rhs.elems[i];
        }
    } else {
        if(elems != NULL) {
            delete[] elems;
        }
    }
}

mesh::~mesh() {
    if(pnts != NULL) {
        delete[] pnts;
    }
    if(elems != NULL) {
        delete[] elems;
    }
    if(chiefPnts != NULL) {
        delete[] chiefPnts;
    }
}

mesh& mesh::operator=(const mesh &rhs) {
    if(rhs.numPnts > 0) {
        if(pnts != NULL) {
            delete[] pnts;
        }
        numPnts = rhs.numPnts;
        pnts = new cartCoord[numPnts];
        for(int i=0;i<numPnts;i++) {
            pnts[i] = rhs.pnts[i];
        }
    } else {
        if(pnts != NULL) {
            delete[] pnts;
        }
    }
    
    if(rhs.numElems > 0) {
        if(elems != NULL) {
            delete[] elems;
        }
        numElems = rhs.numElems;
        elems = new triElem[numElems];
        for(int i=0;i<numPnts;i++) {
            elems[i] = rhs.elems[i];
        }
    } else {
        if(elems != NULL) {
            delete[] elems;
        }
    }
    return *this;
}

void mesh::printBB() {
    cout << "Lower x: " << xl << "Higher x: " << xu << "Lower y: " << yl 
            << "Higher y: " << yu << "Lower z: " << zl << "Higher z: " << zu << endl;
}

ostream& operator<<(ostream &out, const mesh &rhs) {
    for(int i=0;i<rhs.numPnts;i++) {
        cout << rhs.pnts[i] << endl;
    }
    
    for(int i=0;i<rhs.numElems;i++) {
        cout << rhs.elems[i] << endl;
    }
    
    return out;
}

int mesh::findBB(const float threshold) {
    if(numPnts!=0 && numElems!=0) {
        xl = pnts[0].coords[0]; 
        xu = pnts[0].coords[0]; 
        yl = pnts[0].coords[1]; 
        yu = pnts[0].coords[1];
        zl = pnts[0].coords[2]; 
        zu = pnts[0].coords[2];
        for(int i=1;i<numPnts;i++) {
            if(pnts[i].coords[0] < xl) {
                xl = pnts[i].coords[0];
            }
            if(pnts[i].coords[0] > xu) {
                xu = pnts[i].coords[0];
            }
            if(pnts[i].coords[1] < yl) {
                yl = pnts[i].coords[1];
            }
            if(pnts[i].coords[1] > yu) {
                yu = pnts[i].coords[1];
            }
            if(pnts[i].coords[2] < zl) {
                zl = pnts[i].coords[2];
            }
            if(pnts[i].coords[2] > zu) {
                zu = pnts[i].coords[2];
            }
        }
        xl-=threshold;
        xu+=threshold;
        yl-=threshold;
        yu+=threshold;
        zl-=threshold;
        zu+=threshold;
        
        return EXIT_SUCCESS;
    } else {
        cout << "Not enough mesh information!" << endl;
        return EXIT_FAILURE;
    }
}

__global__ void rayTrnglsInt(const cartCoord sp, const cartCoord dir, 
        const cartCoord *pnts, const triElem *elems, const int numElems, bool *flags) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x; 
    if(idx < numElems) {
        flags[idx] = rayTrnglInt(sp,dir,pnts[elems[idx].nodes[0]],pnts[elems[idx].nodes[1]],
                pnts[elems[idx].nodes[2]]);
        //printf("%d: %d\n",idx,flags[idx]);
    }
    
}

__global__ void distPntPnts(const cartCoord sp, const cartCoord *pnts, const int numPnts, float *dists) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x; 
    if(idx < numPnts) {
        cartCoord temp = pnts[idx]-sp;
        dists[idx] = temp.nrm2();
    }
}

int mesh::genCHIEF(const int num, const float threshold) {
    numCHIEF = num;
    if(chiefPnts != NULL) {
        delete[] chiefPnts;
    }
    chiefPnts = new cartCoord[numCHIEF];
    float randNums[3];
    int width = 32, numBlocks;
    float xRand, yRand, zRand;
    unsigned long long seed = time(0);
    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    cartCoord sp;
    cartCoord *pnts_d;
    triElem *elems_d;
    HOST_CALL(meshCloudToGPU(&pnts_d,&elems_d));
    float *dists = new float[numPnts];
    float *dists_d;
    CUDA_CALL(hipMalloc(&dists_d,numPnts*sizeof(float)));
    bool *flags_d;
    CUDA_CALL(hipMalloc(&flags_d,numElems*sizeof(bool)));
    bool *flags = new bool[numElems];
    int cnt = 0; //counter for number of CHIEF points
    while(cnt < numCHIEF) {
        do {
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,seed++));
            CURAND_CALL(hiprandGenerateUniform(gen,randNums,3));
            //printf("(%f,%f,%f)\n",randNums[0],randNums[1],randNums[2]);
            dirCHIEF.set(randNums[0],randNums[1],randNums[2]);
            xRand = descale(xl,xu,randNums[0]);
            yRand = descale(yl,yu,randNums[1]);
            zRand = descale(zl,zu,randNums[2]);
            sp.set(xRand,yRand,zRand);
            numBlocks = (numElems+width-1)/width;
            rayTrnglsInt<<<numBlocks,width>>>(sp,dirCHIEF,pnts_d,elems_d,numElems,flags_d);
            CUDA_CALL(hipMemcpy(flags,flags_d,numElems*sizeof(bool),hipMemcpyDeviceToHost));
            CUDA_CALL(hipDeviceSynchronize());
            numBlocks = (numPnts+width-1)/width;
            distPntPnts<<<numBlocks,width>>>(sp,pnts_d,numPnts,dists_d);
            CUDA_CALL(hipMemcpy(dists,dists_d,numPnts*sizeof(float),hipMemcpyDeviceToHost));
            //printf("Minimum distance: %f\n",arrayMin(dists,numPnts));
            
            //cout << inObj(flags,numElems) << endl;
        } while((!inObj(flags,numElems))||(arrayMin(dists,numPnts)<threshold));
        chiefPnts[cnt].set(xRand,yRand,zRand);
        cnt++;
    }
    
    delete[] dists;
    delete[] flags;
    CUDA_CALL(hipFree(pnts_d));
    CUDA_CALL(hipFree(elems_d));
    CUDA_CALL(hipFree(flags_d));
    CUDA_CALL(hipFree(dists_d));
    CURAND_CALL(hiprandDestroyGenerator(gen));
    
    for(int i=0;i<numCHIEF;i++) {
        cout << chiefPnts[i] << endl;
    }
     
    return EXIT_SUCCESS;
}

int mesh::meshCloudToGPU(cartCoord **pPnts_d,triElem **pElems_d) {
    if(pnts!=NULL && elems!=NULL) {
        CUDA_CALL(hipMalloc(pPnts_d,numPnts*sizeof(cartCoord)));
        CUDA_CALL(hipMemcpy(*pPnts_d,pnts,numPnts*sizeof(cartCoord),hipMemcpyHostToDevice));
        CUDA_CALL(hipMalloc(pElems_d,numElems*sizeof(triElem)));
        CUDA_CALL(hipMemcpy(*pElems_d,elems,numElems*sizeof(triElem),hipMemcpyHostToDevice));
    }
    
    return EXIT_SUCCESS;
}

int mesh::chiefToGPU(cartCoord **pchiefPnts) {
    if(numCHIEF!=0) {
        CUDA_CALL(hipMalloc(pchiefPnts,numCHIEF*sizeof(cartCoord)));
        CUDA_CALL(hipMemcpy(*pchiefPnts,chiefPnts,numCHIEF*sizeof(cartCoord),hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}

int mesh::meshToGPU(cartCoord **pPnts_d, triElem **pElems_d) {
    if(numPnts==0 || numElems==0 || numCHIEF==0) {
        cout << "The mesh object is incomplete." << endl;
        return EXIT_FAILURE;
    } else {
        int i;
        cartCoord *pnts_h = new cartCoord[numPnts+numCHIEF];
        for(i=0;i<numPnts;i++) {
            pnts_h[i] = pnts[i];
        }
        for(i=0;i<numCHIEF;i++) {
            pnts_h[numPnts+i] = chiefPnts[i];
        }
        CUDA_CALL(hipMalloc(pPnts_d,(numPnts+numCHIEF)*sizeof(cartCoord)));
        CUDA_CALL(hipMemcpy(pPnts_d,pnts_h,(numPnts+numCHIEF)*sizeof(cartCoord),hipMemcpyHostToDevice));
        
        CUDA_CALL(hipMalloc(pElems_d,numElems*sizeof(triElem)));
        CUDA_CALL(hipMemcpy(pElems_d,elems,numElems*sizeof(triElem),hipMemcpyHostToDevice));
        return EXIT_SUCCESS;
    }
}

//cartCoord2D
__host__ __device__ cartCoord2D::cartCoord2D(const cartCoord2D &rhs) {
    coords[0] = rhs.coords[0];
    coords[1] = rhs.coords[1];
}

__host__ __device__ cartCoord2D& cartCoord2D::operator=(const cartCoord2D &rhs) {
    coords[0] = rhs.coords[0];
    coords[1] = rhs.coords[1];
    return *this;
}

__host__ __device__ void cartCoord2D::set(const float x, const float y) {
    coords[0] = x;
    coords[1] = y;
}

__host__ __device__ cartCoord2D cartCoord2D::operator+(const cartCoord2D &rhs) const {
    cartCoord2D temp;
    temp.coords[0] = coords[0] + rhs.coords[0];
    temp.coords[1] = coords[1] + rhs.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D cartCoord2D::operator-(const cartCoord2D &rhs) const {
    cartCoord2D temp;
    temp.coords[0] = coords[0] - rhs.coords[0];
    temp.coords[1] = coords[1] - rhs.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D numDvd(const cartCoord2D &dividend, 
        const float divisor) {
    if(divisor == 0) {
        printf("divisor cannot be 0.\n");
        return cartCoord2D(0,0);
    } else {
        return cartCoord2D(dividend.coords[0]/divisor,dividend.coords[1]/divisor);
    }
}

__host__ __device__ cartCoord2D numMul(const float lambda, const cartCoord2D &pnt) {
    return cartCoord2D(lambda*pnt.coords[0],lambda*pnt.coords[1]);
}

__host__ __device__ float Psi_L(const cartCoord pnt) {
    return 1.0/(4*PI*pnt.nrm2());
}

__host__ __device__ float N_1(const cartCoord2D pnt) {
    return pnt.coords[0];
}

__host__ __device__ float N_2(const cartCoord2D pnt) {
    return pnt.coords[1];
}

__host__ __device__ float N_3(const cartCoord2D pnt) {
    return 1-pnt.coords[0]-pnt.coords[1];
}

__host__ __device__ float pN1pXi1(const cartCoord2D pnt) {
    return 1.0;
}

__host__ __device__ float pN1pXi2(const cartCoord2D pnt) {
    return 0;
}

__host__ __device__ float pN2pXi1(const cartCoord2D pnt) {
    return 0;
}

__host__ __device__ float pN2pXi2(const cartCoord2D pnt) {
    return 1.0;
}

__host__ __device__ float pN3pXi1(const cartCoord2D pnt) {
    return -1.0;
}

__host__ __device__ float pN3pXi2(const cartCoord2D pnt) {
    return -1.0;
}

__host__ __device__ cartCoord xiToElem(const cartCoord pnt1, const cartCoord pnt2,
        const cartCoord pnt3, const cartCoord2D localPnt) {
    return numMul(N_1(localPnt),pnt1)+numMul(N_2(localPnt),pnt2)
            +numMul(N_3(localPnt),pnt3); 
}

__host__ __device__ cartCoord pRvpXi1TimespRvpXi2(const cartCoord pnt1, const cartCoord pnt2, 
        const cartCoord pnt3) {
    return (pnt1-pnt3)*(pnt2-pnt3);
}

__host__ __device__ cartCoord2D etaToRhoTheta(const cartCoord2D s) {
    cartCoord2D temp;
    temp.coords[0] = 0.5+0.5*s.coords[0];
    temp.coords[1] = 0.5+0.5*s.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D rhoThetaToXi_3(const cartCoord2D s) {
    cartCoord2D temp;
    temp.coords[0] = s.coords[0]*(1-s.coords[1]); //rho*(1-theta)
    temp.coords[1] = s.coords[0]*s.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D rhoThetaToXi_1(const cartCoord2D s) {
    cartCoord2D temp;
    temp.coords[0] = 1-s.coords[0];
    temp.coords[1] = s.coords[0]*(1-s.coords[1]);
    return temp;
}

__host__ __device__ cartCoord2D rhoThetaToXi_2(const cartCoord2D s) {
    cartCoord2D  temp;
    temp.coords[0] = s.coords[0]*s.coords[1];
    temp.coords[1] = 1-s.coords[0];
    return temp;
}

