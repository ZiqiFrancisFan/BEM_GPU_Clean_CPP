#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

#include <hip/hip_complex.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h>
#include<ctime>
#include "mesh.h"
#include "numerical.h"

//cartCoord class functions
__host__ __device__ cartCoord::cartCoord(const cartCoord &rhs) {
    for(int i=0;i<3;i++) {
        coords[i] = rhs.coords[i];
    }
}

__host__ __device__ void cartCoord::set(const float x, const float y, const float z) {
    coords[0] = x;
    coords[1] = y;
    coords[2] = z;
}

ostream& operator<<(ostream &out,const cartCoord &rhs) {
    out << "(" << rhs.coords[0] << "," << rhs.coords[1] << "," << rhs.coords[2] 
            << ")";
    return out;
}

__host__ __device__ cartCoord& cartCoord::operator=(const cartCoord &rhs) {
    coords[0] = rhs.coords[0];
    coords[1] = rhs.coords[1];
    coords[2] = rhs.coords[2];
    return *this;
}

__host__ __device__ cartCoord cartCoord::operator+(const cartCoord &rhs) const {
    cartCoord temp;
    temp.coords[0] = coords[0] + rhs.coords[0];
    temp.coords[1] = coords[1] + rhs.coords[1];
    temp.coords[2] = coords[2] + rhs.coords[2];
    return temp;
}

__host__ __device__ cartCoord cartCoord::operator-(const cartCoord &rhs) const {
    cartCoord temp;
    temp.coords[0] = coords[0] - rhs.coords[0];
    temp.coords[1] = coords[1] - rhs.coords[1];
    temp.coords[2] = coords[2] - rhs.coords[2];
    return temp;
}

__host__ __device__ cartCoord cartCoord::operator-() const {
    return cartCoord(-coords[0],-coords[1],-coords[2]);
}

__host__ __device__ cartCoord cartCoord::operator*(const cartCoord &rhs) const {
    cartCoord prod;
    prod.coords[0] = coords[1]*rhs.coords[2]-coords[2]*rhs.coords[1];
    prod.coords[1] = coords[2]*rhs.coords[0]-coords[0]*rhs.coords[2];
    prod.coords[2] = coords[0]*rhs.coords[1]-coords[1]*rhs.coords[0];
    return prod;
}

__host__ __device__ float dotProd(const cartCoord &p1,const cartCoord &p2) {
    return p1.coords[0]*p2.coords[0]+p1.coords[1]*p2.coords[1]+p1.coords[2]*p2.coords[2];
}

__host__ __device__ cartCoord numDvd(const cartCoord &pnt, const float lambda) {
    if(lambda == 0) {
        printf("divisor cannot be 0.\n");
        return cartCoord(0,0,0);
    } else {
        return cartCoord(pnt.coords[0]/lambda,pnt.coords[1]/lambda,pnt.coords[2]/lambda);
    }
}

__host__ __device__ cartCoord numMul(const float lambda, const cartCoord &pnt) {
    return cartCoord(lambda*pnt.coords[0],lambda*pnt.coords[1],lambda*pnt.coords[2]);
}

__host__ __device__ float cartCoord::nrm2() const {
    return sqrtf(powf(coords[0],2)+powf(coords[1],2)+powf(coords[2],2));
}

__host__ __device__ float r(const cartCoord p1, const cartCoord p2) {
    cartCoord temp = p1-p2;
    return temp.nrm2();
}

__host__ __device__ float prpn2(const cartCoord n, const cartCoord p1, const cartCoord p2) {
    return ((p1.coords[0]-p2.coords[0])*n.coords[0]+(p1.coords[1]-p2.coords[1])*n.coords[1]
            +(p1.coords[2]-p2.coords[2])*n.coords[2])/r(p1,p2);
}

__host__ __device__ float prRpn2(const cartCoord n, const cartCoord p1, const cartCoord p2) {
    float temp1 = 1.0/powf(r(p1,p2),2), temp2 = prpn2(n,p1,p2);
    return -temp1*temp2;
}

__host__ __device__ hipFloatComplex green2(const float k, const cartCoord x, const cartCoord y) {
    cartCoord temp = x-y;
    float r = temp.nrm2();
    return green(k,r);
}

__host__ __device__ float PsiL2(const cartCoord p1, const cartCoord p2) {
    cartCoord temp = p1-p2;
    return PsiL(temp.nrm2());
}

__host__ __device__ float pPsiLpn2(const cartCoord n, const cartCoord p1, 
        const cartCoord p2) {
    return 1.0/(4*PI)*prRpn2(n,p1,p2);
}

__host__ __device__ hipFloatComplex pGpn2(const float k, const cartCoord n, 
        const cartCoord p1, const cartCoord p2) {
    hipFloatComplex temp1 = green2(k,p1,p2), temp2 = make_hipFloatComplex(-1.0/r(p1,p2),k);
    hipFloatComplex temp3 = hipCmulf(temp1,temp2);
    float temp4 = prpn2(n,p1,p2);
    return make_hipFloatComplex(temp4*hipCrealf(temp3),temp4*hipCimagf(temp3));
}

__device__ hipFloatComplex g_l1_nsgl(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g)); //the imag part
}

__device__ hipFloatComplex g_l2_nsgl(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g)); //the imag part
}

__device__ hipFloatComplex g_l3_nsgl(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g)); //the imag part
}

__device__ hipFloatComplex h_l1_nsgl(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l2_nsgl(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
} 

__device__ hipFloatComplex h_l3_nsgl(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
} 

__device__ hipFloatComplex g_l1_sgl1(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l2_sgl1(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l3_sgl1(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l1_sgl2(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l2_sgl2(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l3_sgl2(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l1_sgl3(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l2_sgl3(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex g_l3_sgl3(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp, omega = k*speed;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, crossProd;
    hipFloatComplex g;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    g = green2(k,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd*density*omega;
    return make_hipFloatComplex(-temp*hipCimagf(g),temp*hipCrealf(g));
}

__device__ hipFloatComplex h_l1_sgl1(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l2_sgl1(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l3_sgl1(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l1_sgl2(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l2_sgl2(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l3_sgl2(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l1_sgl3(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_1(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l2_sgl3(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_2(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ hipFloatComplex h_l3_sgl3(const float k, const cartCoord x, 
        const cartCoord p1, const cartCoord p2, const cartCoord p3, 
        const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, N, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    hipFloatComplex gDrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    N = N_3(cartCoord2D(xi1,xi2));
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    gDrv = pGpn2(k,nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*N*vertCrossProd;
    return make_hipFloatComplex(temp*hipCrealf(gDrv),temp*hipCimagf(gDrv));
}

__device__ float c_l_nsgl(const cartCoord x, const cartCoord p1, const cartCoord p2, 
        const cartCoord p3, const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    float psiLdrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    psiLdrv = pPsiLpn2(nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*psiLdrv;
    return temp;
}

__device__ float c_l_sgl1(const cartCoord x, const cartCoord p1, const cartCoord p2, 
        const cartCoord p3, const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    float psiLdrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = 1-rho;
    xi2 = rho*(1-theta);
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    psiLdrv = pPsiLpn2(nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*psiLdrv;
    return temp;
}

__device__ float c_l_sgl2(const cartCoord x, const cartCoord p1, const cartCoord p2, 
        const cartCoord p3, const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    float psiLdrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*theta;
    xi2 = 1-rho;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    psiLdrv = pPsiLpn2(nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*psiLdrv;
    return temp;
}

__device__ float c_l_sgl3(const cartCoord x, const cartCoord p1, const cartCoord p2, 
        const cartCoord p3, const int n, const int m) {
    float xi1, xi2, rho, theta, vertCrossProd, temp;
    float eta1 = INTPNTS[m], eta2 = INTPNTS[n];
    cartCoord y, nrml, crossProd;
    float psiLdrv;
    rho = 0.5+0.5*eta1;
    theta = 0.5+0.5*eta2;
    xi1 = rho*(1-theta);
    xi2 = rho*theta;
    crossProd = (p1-p3)*(p2-p3);
    vertCrossProd = crossProd.nrm2();
    nrml = crossProd.nrmlzd();
    y = xiToElem(p1,p2,p3,cartCoord2D(xi1,xi2));
    psiLdrv = pPsiLpn2(nrml,x,y);
    temp = 0.25*INTWGTS[n]*INTWGTS[m]*rho*vertCrossProd*psiLdrv;
    return temp;
}

//hCoeffs is of size 3*numPnts and cCoeffs is of size numPnts
__global__ void pntsElem_lnm_nsgl(const float k, const int l, const int n, const int m, 
        const triElem *elems, const cartCoord *pnts, const int numPnts, 
        hipFloatComplex *hCoeffs, hipFloatComplex *gCoeffs, float *cCoeffs) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numPnts) {
        triElem elem = elems[l]; //get the current element
  
        hCoeffs[3*idx] = h_l1_nsgl(k,pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs[3*idx+1] = h_l2_nsgl(k,pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs[3*idx+2] = h_l3_nsgl(k,pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        gCoeffs[3*idx] = g_l1_nsgl(k,pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs[3*idx+1] = g_l2_nsgl(k,pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs[3*idx+2] = g_l3_nsgl(k,pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        cCoeffs[idx] = c_l_nsgl(pnts[idx],pnts[elem.nodes[0]],pnts[elem.nodes[1]],pnts[elem.nodes[2]],
                n,m);
    }
    //Singularity integral has to be eliminated on the CPU end!
}

//gCoeffs and hCoeffs are of size 3*numElems, cCoeffs is of size numElems
__global__ void pntsElems_nm_sgl(const float k, const int n, const int m, const triElem *elems, 
        const int numElems, const cartCoord *pnts, hipFloatComplex *hCoeffs_sgl1, 
        hipFloatComplex *hCoeffs_sgl2, hipFloatComplex *hCoeffs_sgl3, hipFloatComplex *gCoeffs_sgl1, 
        hipFloatComplex *gCoeffs_sgl2, hipFloatComplex *gCoeffs_sgl3, float *cCoeffs_sgl1, 
        float *cCoeffs_sgl2, float *cCoeffs_sgl3) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElems) {
        triElem elem = elems[idx];
        
        hCoeffs_sgl1[3*idx] = h_l1_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs_sgl1[3*idx+1] = h_l2_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs_sgl1[3*idx+2] = h_l3_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        hCoeffs_sgl2[3*idx] = h_l1_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs_sgl2[3*idx+1] = h_l2_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs_sgl2[3*idx+2] = h_l3_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        hCoeffs_sgl3[3*idx] = h_l1_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs_sgl3[3*idx+1] = h_l2_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        hCoeffs_sgl3[3*idx+2] = h_l3_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        gCoeffs_sgl1[3*idx] = g_l1_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs_sgl1[3*idx+1] = g_l2_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs_sgl1[3*idx+2] = g_l3_sgl1(k,pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        gCoeffs_sgl2[3*idx] = g_l1_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs_sgl2[3*idx+1] = g_l2_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs_sgl2[3*idx+2] = g_l3_sgl2(k,pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        gCoeffs_sgl3[3*idx] = g_l1_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs_sgl3[3*idx+1] = g_l2_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        gCoeffs_sgl3[3*idx+2] = g_l3_sgl3(k,pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        cCoeffs_sgl1[idx] = c_l_sgl1(pnts[elem.nodes[0]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        cCoeffs_sgl2[idx] = c_l_sgl2(pnts[elem.nodes[1]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
        
        cCoeffs_sgl3[idx] = c_l_sgl3(pnts[elem.nodes[2]],pnts[elem.nodes[0]],pnts[elem.nodes[1]],
                pnts[elem.nodes[2]],n,m);
    }
}

//Updated in an n and m loop
__global__ void updateSystemLhs_hg_nsgl(hipFloatComplex *A, const int numPnts, const int numCHIEF, 
        const int lda, const hipFloatComplex *hCoeffs, const hipFloatComplex *gCoeffs, 
        const triElem *elems, const int l) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numPnts+numCHIEF) {
        triElem elem = elems[l];
        hipFloatComplex hContrs[3], gContrs[3]; 
        hipFloatComplex pContrs[3], bc;
        
        bc = hipCdivf(elem.bc[0],elem.bc[1]);
        
        hContrs[0] = hCoeffs[3*idx];
        hContrs[1] = hCoeffs[3*idx+1];
        hContrs[2] = hCoeffs[3*idx+2];
        
        gContrs[0] = gCoeffs[3*idx];
        gContrs[1] = gCoeffs[3*idx+1];
        gContrs[2] = gCoeffs[3*idx+2];
        
        pContrs[0] = hipCsubf(hContrs[0],hipCmulf(bc,gContrs[0]));
        pContrs[1] = hipCsubf(hContrs[1],hipCmulf(bc,gContrs[1]));
        pContrs[2] = hipCsubf(hContrs[2],hipCmulf(bc,gContrs[2]));
        
        A[IDXC0(idx,elem.nodes[0],lda)] = hipCaddf(A[IDXC0(idx,elem.nodes[0],lda)],pContrs[0]);
        A[IDXC0(idx,elem.nodes[1],lda)] = hipCaddf(A[IDXC0(idx,elem.nodes[1],lda)],pContrs[1]);
        A[IDXC0(idx,elem.nodes[2],lda)] = hipCaddf(A[IDXC0(idx,elem.nodes[2],lda)],pContrs[2]);
    }
}

//numPnts does not include CHIEF points 
//Updated in an n and m loop
__global__ void updateSystemLhs_c_nsgl(hipFloatComplex *A, const int numPnts, const int lda, 
        const float *cCoeffs) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numPnts) {
        A[IDXC0(idx,idx,lda)] = hipCsubf(A[IDXC0(idx,idx,lda)],make_hipFloatComplex(cCoeffs[idx],0));
    }
}

//Updated in an n and m loop
__global__ void updateSystemRhs_nsgl(hipFloatComplex *B, const int numPnts, const int numCHIEF, 
        const int ldb, const int srcIdx, const hipFloatComplex *gCoeffs, const triElem *elems, 
        const int l) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numPnts+numCHIEF) {
        hipFloatComplex gContrs[3], bc, temp;
        gContrs[0] = gCoeffs[3*idx];
        gContrs[1] = gCoeffs[3*idx+1];
        gContrs[2] = gCoeffs[3*idx+2];
        
        triElem elem = elems[l];
        bc = hipCdivf(elem.bc[2],elem.bc[1]);
        
        temp = hipCmulf(bc,gContrs[0]);
        B[IDXC0(idx,srcIdx,ldb)] = hipCsubf(B[IDXC0(idx,srcIdx,ldb)],temp);
        temp = hipCmulf(bc,gContrs[1]);
        B[IDXC0(idx,srcIdx,ldb)] = hipCsubf(B[IDXC0(idx,srcIdx,ldb)],temp);
        temp = hipCmulf(bc,gContrs[2]);
        B[IDXC0(idx,srcIdx,ldb)] = hipCsubf(B[IDXC0(idx,srcIdx,ldb)],temp);
    }
}


//Updated in an n and m loop
__global__ void updateSystemLhs_hg_sgl(hipFloatComplex *A, const int lda, 
        hipFloatComplex *hCoeffs_sgl1, hipFloatComplex *hCoeffs_sgl2, hipFloatComplex *hCoeffs_sgl3, 
        hipFloatComplex *gCoeffs_sgl1, hipFloatComplex *gCoeffs_sgl2, hipFloatComplex *gCoeffs_sgl3, 
        const triElem *elems, const int numElems) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElems) {
        triElem elem = elems[idx];
        hipFloatComplex hContrs_sgl1[3], hContrs_sgl2[3], hContrs_sgl3[3], 
                gContrs_sgl1[3], gContrs_sgl2[3], gContrs_sgl3[3], 
                pContrs_sgl1[3], pContrs_sgl2[3], pContrs_sgl3[3];
        hipFloatComplex temp, bc = hipCdivf(elem.bc[0],elem.bc[1]);
        
        //singular 1:
        hContrs_sgl1[0] = hCoeffs_sgl1[3*idx];
        hContrs_sgl1[1] = hCoeffs_sgl1[3*idx+1];
        hContrs_sgl1[2] = hCoeffs_sgl1[3*idx+2];
        
        gContrs_sgl1[0] = gCoeffs_sgl1[3*idx];
        gContrs_sgl1[1] = gCoeffs_sgl1[3*idx+1];
        gContrs_sgl1[2] = gCoeffs_sgl1[3*idx+2];
        
        temp = hipCmulf(bc,gContrs_sgl1[0]);
        pContrs_sgl1[0] = hipCsubf(hContrs_sgl1[0],temp);
        temp = hipCmulf(bc,gContrs_sgl1[1]);
        pContrs_sgl1[1] = hipCsubf(hContrs_sgl1[1],temp);
        temp = hipCmulf(bc,gContrs_sgl1[2]);
        pContrs_sgl1[2] = hipCsubf(hContrs_sgl1[2],temp);
        
        A[IDXC0(elem.nodes[0],elem.nodes[0],lda)] = hipCaddf(A[IDXC0(elem.nodes[0],elem.nodes[0],lda)],
                pContrs_sgl1[0]);
        A[IDXC0(elem.nodes[0],elem.nodes[1],lda)] = hipCaddf(A[IDXC0(elem.nodes[0],elem.nodes[1],lda)],
                pContrs_sgl1[1]);
        A[IDXC0(elem.nodes[0],elem.nodes[2],lda)] = hipCaddf(A[IDXC0(elem.nodes[0],elem.nodes[2],lda)],
                pContrs_sgl1[2]);
        
        //singular 2:
        hContrs_sgl2[0] = hCoeffs_sgl2[3*idx];
        hContrs_sgl2[1] = hCoeffs_sgl2[3*idx+1];
        hContrs_sgl2[2] = hCoeffs_sgl2[3*idx+2];
        
        gContrs_sgl2[0] = gCoeffs_sgl2[3*idx];
        gContrs_sgl2[1] = gCoeffs_sgl2[3*idx+1];
        gContrs_sgl2[2] = gCoeffs_sgl2[3*idx+2];
        
        temp = hipCmulf(bc,gContrs_sgl2[0]);
        pContrs_sgl2[0] = hipCsubf(hContrs_sgl2[0],temp);
        temp = hipCmulf(bc,gContrs_sgl2[1]);
        pContrs_sgl2[1] = hipCsubf(hContrs_sgl2[1],temp);
        temp = hipCmulf(bc,gContrs_sgl1[2]);
        pContrs_sgl2[2] = hipCsubf(hContrs_sgl2[2],temp);
        
        A[IDXC0(elem.nodes[1],elem.nodes[0],lda)] = hipCaddf(A[IDXC0(elem.nodes[1],elem.nodes[0],lda)],
                pContrs_sgl2[0]);
        A[IDXC0(elem.nodes[1],elem.nodes[1],lda)] = hipCaddf(A[IDXC0(elem.nodes[1],elem.nodes[1],lda)],
                pContrs_sgl2[1]);
        A[IDXC0(elem.nodes[1],elem.nodes[2],lda)] = hipCaddf(A[IDXC0(elem.nodes[1],elem.nodes[2],lda)],
                pContrs_sgl2[2]);
        
        //singular 2:
        hContrs_sgl3[0] = hCoeffs_sgl3[3*idx];
        hContrs_sgl3[1] = hCoeffs_sgl3[3*idx+1];
        hContrs_sgl3[2] = hCoeffs_sgl3[3*idx+2];
        
        gContrs_sgl3[0] = gCoeffs_sgl3[3*idx];
        gContrs_sgl3[1] = gCoeffs_sgl3[3*idx+1];
        gContrs_sgl3[2] = gCoeffs_sgl3[3*idx+2];
        
        temp = hipCmulf(bc,gContrs_sgl3[0]);
        pContrs_sgl3[0] = hipCsubf(hContrs_sgl3[0],temp);
        temp = hipCmulf(bc,gContrs_sgl3[1]);
        pContrs_sgl3[1] = hipCsubf(hContrs_sgl3[1],temp);
        temp = hipCmulf(bc,gContrs_sgl3[2]);
        pContrs_sgl3[2] = hipCsubf(hContrs_sgl3[2],temp);
        
        A[IDXC0(elem.nodes[2],elem.nodes[0],lda)] = hipCaddf(A[IDXC0(elem.nodes[2],elem.nodes[0],lda)],
                pContrs_sgl3[0]);
        A[IDXC0(elem.nodes[2],elem.nodes[1],lda)] = hipCaddf(A[IDXC0(elem.nodes[2],elem.nodes[1],lda)],
                pContrs_sgl3[1]);
        A[IDXC0(elem.nodes[2],elem.nodes[2],lda)] = hipCaddf(A[IDXC0(elem.nodes[2],elem.nodes[2],lda)],
                pContrs_sgl3[2]);
    }
}

__global__ void updateSystemLhs_c_sgl(hipFloatComplex *A, const int lda, float *cCoeffs_sgl1, 
        float *cCoeffs_sgl2, float *cCoeffs_sgl3, const triElem *elems, const int numElems) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElems) {
        triElem elem = elems[idx];
        float cContr_sgl1, cContr_sgl2, cContr_sgl3;
        
        cContr_sgl1 = cCoeffs_sgl1[idx];
        A[IDXC0(elem.nodes[0],elem.nodes[0],lda)] = hipCsubf(A[IDXC0(elem.nodes[0],elem.nodes[0],lda)],
                make_hipFloatComplex(cContr_sgl1,0));
        
        cContr_sgl2 = cCoeffs_sgl2[idx];
        A[IDXC0(elem.nodes[1],elem.nodes[1],lda)] = hipCsubf(A[IDXC0(elem.nodes[1],elem.nodes[1],lda)],
                make_hipFloatComplex(cContr_sgl2,0));
        
        cContr_sgl3 = cCoeffs_sgl3[idx];
        A[IDXC0(elem.nodes[2],elem.nodes[2],lda)] = hipCsubf(A[IDXC0(elem.nodes[2],elem.nodes[2],lda)],
                make_hipFloatComplex(cContr_sgl3,0));
    }
}



__host__ __device__ float trnglArea(const cartCoord p1, const cartCoord p2) {
    cartCoord temp = p1*p2;
    return temp.nrm2()/2.0;
}

__host__ __device__ cartCoord cartCoord::nrmlzd() {
    if(nrm2() == 0) {
        return cartCoord(nanf(""),nanf(""),nanf(""));
    } else {
        float nrm = nrm2();
        return numDvd(*this,nrm);
    }
}

__host__ __device__ cartCoord rayPlaneInt(const cartCoord sp, const cartCoord dir,
    const cartCoord n, const cartCoord pnt) {
    cartCoord temp;
    if(dotProd(n,sp-pnt) == 0) {
        temp =  sp;
    } else {
        if(dotProd(n,dir)==0) {
            temp =  cartCoord(nanf(""),nanf(""),nanf(""));
        } else {
            float t = (dotProd(n,pnt)-dotProd(n,sp))/dotProd(n,dir);
            if(t>0) {
                temp = sp+numMul(t,dir);
            } else {
                temp = cartCoord(nanf(""),nanf(""),nanf(""));
            }
        }
    }
    //printf("(%f,%f,%f)\n",temp.coords[0],temp.coords[1],temp.coords[2]);
    return temp;
}

__host__ __device__ bool cartCoord::isLegal() const {
    if(coords[0]!=coords[0]||coords[1]!=coords[1]||coords[2]!=coords[2]) {
        return false;
    } else {
        return true;
    }
}

__host__ __device__ bool cartCoord::isEqual(const cartCoord p) const {
    if(abs(coords[0]-p.coords[0])<EPS && abs(coords[1]-p.coords[1])<EPS 
            && abs(coords[2]-p.coords[2])<EPS) {
        return true;
    } else {
        return false;
    }
}


__host__ __device__ bool cartCoord::isInsideTrngl(const cartCoord p1,
        const cartCoord p2, const cartCoord p3) const {
    if(!isLegal()) {
        return false;
    } else {
        cartCoord v12 = p2-p1, v23 = p3-p2, v31 = p1-p3;
        cartCoord v1p = *this-p1, v2p = *this-p2, v3p = *this-p3;
        cartCoord t1 = v12*v1p, t2 = v23*v2p, t3 = v31*v3p;
        cartCoord t1Nrm = t1.nrmlzd(), t2Nrm = t2.nrmlzd(), t3Nrm = t3.nrmlzd();
        if(t1Nrm.isEqual(t2Nrm) && t2Nrm.isEqual(t3Nrm)) {
            return true;
        } else {
            return false;
        }
    }
}

__host__ __device__ bool rayTrnglInt(const cartCoord sp, const cartCoord dir,
    const cartCoord p1, const cartCoord p2, const cartCoord p3) {
    cartCoord n = (p2-p1)*(p3-p1);
    cartCoord intPnt = rayPlaneInt(sp,dir,n,p1);
    return intPnt.isInsideTrngl(p1,p2,p3);
}

//triangular element class
__host__ __device__ triElem::triElem(const triElem &rhs) {
    for(int i=0;i<3;i++) {
        nodes[i] = rhs.nodes[i];
        bc[i] = rhs.bc[i];
    }
}

__host__ __device__ triElem& triElem::operator=(const triElem &rhs) {
    for(int i=0;i<3;i++) {
        nodes[i] = rhs.nodes[i];
        bc[i] = rhs.bc[i];
    }
    return *this;
}

ostream& operator<<(ostream &out, const triElem &rhs) {
    out << "nodes indices: " << rhs.nodes[0] << ", " << rhs.nodes[1] << ", " 
            << rhs.nodes[2] << " boundary condition: " << "A = " << rhs.bc[0] 
            << ", B = " << rhs.bc[1] << ", C = " << rhs.bc[2];
    return out;
}

//class mesh
int mesh::readObj(const char *file) {
    int temp[3];
    vector<cartCoord> pntVec; 
    vector<triElem> elemVec;
    cartCoord pnt;
    triElem elem;
    FILE *fp = fopen(file,"r");
    if (fp == NULL) {
        printf("Failed to open file.\n");
        return EXIT_FAILURE;
    }
    int i = 0;
    char line[50];
    char type[5];
    while (fgets(line,49,fp) != NULL) {
        if (line[0] == 'v') {
            sscanf(line,"%s %f %f %f",type,&pnt.coords[0],&pnt.coords[1],&pnt.coords[2]);
            pntVec.push_back(pnt);
        }

        if (line[0] == 'f') {
            sscanf(line,"%s %d %d %d",type,&temp[0],&temp[1],&temp[2]);
            elem.nodes[0] = temp[0]-1;
            elem.nodes[1] = temp[1]-1;
            elem.nodes[2] = temp[2]-1;
            elem.bc[0] = make_hipFloatComplex(0,0); // ca=0
            elem.bc[1] = make_hipFloatComplex(1,0); // cb=1
            elem.bc[2] = make_hipFloatComplex(0,0); // cc=0
            elemVec.push_back(elem);
        }
    }
    if(pnts != NULL) {
        delete[] pnts;
    }
    pnts = new cartCoord[pntVec.size()];
    for(i=0;i<pntVec.size();i++) {
        pnts[i] = pntVec[i];
    }
    numPnts = pntVec.size();
    
    if(elems != NULL) {
        delete[] elems;
    }
    elems = new triElem[elemVec.size()];
    for(i=0;i<elemVec.size();i++) {
        elems[i] = elemVec[i];
    }
    numElems = elemVec.size();
    fclose(fp);
    return EXIT_SUCCESS;
}

mesh::mesh(const mesh &rhs) {
    if(rhs.numPnts > 0) {
        if(pnts != NULL) {
            delete[] pnts;
        }
        numPnts = rhs.numPnts;
        pnts = new cartCoord[numPnts];
        for(int i=0;i<numPnts;i++) {
            pnts[i] = rhs.pnts[i];
        }
    } else {
        if(pnts != NULL) {
            delete[] pnts;
        }
    }
    
    if(rhs.numElems > 0) {
        if(elems != NULL) {
            delete[] elems;
        }
        numElems = rhs.numElems;
        elems = new triElem[numElems];
        for(int i=0;i<numPnts;i++) {
            elems[i] = rhs.elems[i];
        }
    } else {
        if(elems != NULL) {
            delete[] elems;
        }
    }
}

mesh::~mesh() {
    if(pnts != NULL) {
        delete[] pnts;
    }
    if(elems != NULL) {
        delete[] elems;
    }
    if(chiefPnts != NULL) {
        delete[] chiefPnts;
    }
}

mesh& mesh::operator=(const mesh &rhs) {
    if(rhs.numPnts > 0) {
        if(pnts != NULL) {
            delete[] pnts;
        }
        numPnts = rhs.numPnts;
        pnts = new cartCoord[numPnts];
        for(int i=0;i<numPnts;i++) {
            pnts[i] = rhs.pnts[i];
        }
    } else {
        if(pnts != NULL) {
            delete[] pnts;
        }
    }
    
    if(rhs.numElems > 0) {
        if(elems != NULL) {
            delete[] elems;
        }
        numElems = rhs.numElems;
        elems = new triElem[numElems];
        for(int i=0;i<numPnts;i++) {
            elems[i] = rhs.elems[i];
        }
    } else {
        if(elems != NULL) {
            delete[] elems;
        }
    }
    return *this;
}

void mesh::printBB() {
    cout << "Lower x: " << xl << "Higher x: " << xu << "Lower y: " << yl 
            << "Higher y: " << yu << "Lower z: " << zl << "Higher z: " << zu << endl;
}

ostream& operator<<(ostream &out, const mesh &rhs) {
    for(int i=0;i<rhs.numPnts;i++) {
        cout << rhs.pnts[i] << endl;
    }
    
    for(int i=0;i<rhs.numElems;i++) {
        cout << rhs.elems[i] << endl;
    }
    
    return out;
}

int mesh::findBB(const float threshold) {
    if(numPnts!=0 && numElems!=0) {
        xl = pnts[0].coords[0]; 
        xu = pnts[0].coords[0]; 
        yl = pnts[0].coords[1]; 
        yu = pnts[0].coords[1];
        zl = pnts[0].coords[2]; 
        zu = pnts[0].coords[2];
        for(int i=1;i<numPnts;i++) {
            if(pnts[i].coords[0] < xl) {
                xl = pnts[i].coords[0];
            }
            if(pnts[i].coords[0] > xu) {
                xu = pnts[i].coords[0];
            }
            if(pnts[i].coords[1] < yl) {
                yl = pnts[i].coords[1];
            }
            if(pnts[i].coords[1] > yu) {
                yu = pnts[i].coords[1];
            }
            if(pnts[i].coords[2] < zl) {
                zl = pnts[i].coords[2];
            }
            if(pnts[i].coords[2] > zu) {
                zu = pnts[i].coords[2];
            }
        }
        xl-=threshold;
        xu+=threshold;
        yl-=threshold;
        yu+=threshold;
        zl-=threshold;
        zu+=threshold;
        
        return EXIT_SUCCESS;
    } else {
        cout << "Not enough mesh information!" << endl;
        return EXIT_FAILURE;
    }
}

__global__ void rayTrnglsInt(const cartCoord sp, const cartCoord dir, 
        const cartCoord *pnts, const triElem *elems, const int numElems, bool *flags) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x; 
    if(idx < numElems) {
        flags[idx] = rayTrnglInt(sp,dir,pnts[elems[idx].nodes[0]],pnts[elems[idx].nodes[1]],
                pnts[elems[idx].nodes[2]]);
        //printf("%d: %d\n",idx,flags[idx]);
    }
    
}

__global__ void distPntPnts(const cartCoord sp, const cartCoord *pnts, const int numPnts, float *dists) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x; 
    if(idx < numPnts) {
        cartCoord temp = pnts[idx]-sp;
        dists[idx] = temp.nrm2();
    }
}

int mesh::genCHIEF(const int num, const float threshold) {
    numCHIEF = num;
    if(chiefPnts != NULL) {
        delete[] chiefPnts;
    }
    chiefPnts = new cartCoord[numCHIEF];
    float randNums[3];
    int width = 32, numBlocks;
    float xRand, yRand, zRand;
    unsigned long long seed = time(0);
    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    cartCoord sp;
    cartCoord *pnts_d;
    triElem *elems_d;
    HOST_CALL(meshCloudToGPU(&pnts_d,&elems_d));
    float *dists = new float[numPnts];
    float *dists_d;
    CUDA_CALL(hipMalloc(&dists_d,numPnts*sizeof(float)));
    bool *flags_d;
    CUDA_CALL(hipMalloc(&flags_d,numElems*sizeof(bool)));
    bool *flags = new bool[numElems];
    int cnt = 0; //counter for number of CHIEF points
    while(cnt < numCHIEF) {
        do {
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,seed++));
            CURAND_CALL(hiprandGenerateUniform(gen,randNums,3));
            //printf("(%f,%f,%f)\n",randNums[0],randNums[1],randNums[2]);
            dirCHIEF.set(randNums[0],randNums[1],randNums[2]);
            xRand = descale(xl,xu,randNums[0]);
            yRand = descale(yl,yu,randNums[1]);
            zRand = descale(zl,zu,randNums[2]);
            sp.set(xRand,yRand,zRand);
            numBlocks = (numElems+width-1)/width;
            rayTrnglsInt<<<numBlocks,width>>>(sp,dirCHIEF,pnts_d,elems_d,numElems,flags_d);
            CUDA_CALL(hipMemcpy(flags,flags_d,numElems*sizeof(bool),hipMemcpyDeviceToHost));
            CUDA_CALL(hipDeviceSynchronize());
            numBlocks = (numPnts+width-1)/width;
            distPntPnts<<<numBlocks,width>>>(sp,pnts_d,numPnts,dists_d);
            CUDA_CALL(hipMemcpy(dists,dists_d,numPnts*sizeof(float),hipMemcpyDeviceToHost));
            //printf("Minimum distance: %f\n",arrayMin(dists,numPnts));
            
            //cout << inObj(flags,numElems) << endl;
        } while((!inObj(flags,numElems))||(arrayMin(dists,numPnts)<threshold));
        chiefPnts[cnt].set(xRand,yRand,zRand);
        cnt++;
    }
    
    delete[] dists;
    delete[] flags;
    CUDA_CALL(hipFree(pnts_d));
    CUDA_CALL(hipFree(elems_d));
    CUDA_CALL(hipFree(flags_d));
    CUDA_CALL(hipFree(dists_d));
    CURAND_CALL(hiprandDestroyGenerator(gen));
    
    for(int i=0;i<numCHIEF;i++) {
        cout << chiefPnts[i] << endl;
    }
     
    return EXIT_SUCCESS;
}

int mesh::meshCloudToGPU(cartCoord **pPnts_d,triElem **pElems_d) {
    if(pnts!=NULL && elems!=NULL) {
        CUDA_CALL(hipMalloc(pPnts_d,numPnts*sizeof(cartCoord)));
        CUDA_CALL(hipMemcpy(*pPnts_d,pnts,numPnts*sizeof(cartCoord),hipMemcpyHostToDevice));
        CUDA_CALL(hipMalloc(pElems_d,numElems*sizeof(triElem)));
        CUDA_CALL(hipMemcpy(*pElems_d,elems,numElems*sizeof(triElem),hipMemcpyHostToDevice));
    }
    
    return EXIT_SUCCESS;
}

int mesh::chiefToGPU(cartCoord **pchiefPnts) {
    if(numCHIEF!=0) {
        CUDA_CALL(hipMalloc(pchiefPnts,numCHIEF*sizeof(cartCoord)));
        CUDA_CALL(hipMemcpy(*pchiefPnts,chiefPnts,numCHIEF*sizeof(cartCoord),hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}

int mesh::meshToGPU(cartCoord **pPnts_d, triElem **pElems_d) {
    if(numPnts==0 || numElems==0 || numCHIEF==0) {
        cout << "The mesh object is incomplete." << endl;
        return EXIT_FAILURE;
    } else {
        int i;
        cartCoord *pnts_h = new cartCoord[numPnts+numCHIEF];
        for(i=0;i<numPnts;i++) {
            pnts_h[i] = pnts[i];
        }
        for(i=0;i<numCHIEF;i++) {
            pnts_h[numPnts+i] = chiefPnts[i];
        }
        CUDA_CALL(hipMalloc(pPnts_d,(numPnts+numCHIEF)*sizeof(cartCoord)));
        CUDA_CALL(hipMemcpy(pPnts_d,pnts_h,(numPnts+numCHIEF)*sizeof(cartCoord),hipMemcpyHostToDevice));
        
        CUDA_CALL(hipMalloc(pElems_d,numElems*sizeof(triElem)));
        CUDA_CALL(hipMemcpy(pElems_d,elems,numElems*sizeof(triElem),hipMemcpyHostToDevice));
        return EXIT_SUCCESS;
    }
}

//cartCoord2D
__host__ __device__ cartCoord2D::cartCoord2D(const cartCoord2D &rhs) {
    coords[0] = rhs.coords[0];
    coords[1] = rhs.coords[1];
}

__host__ __device__ cartCoord2D& cartCoord2D::operator=(const cartCoord2D &rhs) {
    coords[0] = rhs.coords[0];
    coords[1] = rhs.coords[1];
    return *this;
}

__host__ __device__ void cartCoord2D::set(const float x, const float y) {
    coords[0] = x;
    coords[1] = y;
}

__host__ __device__ cartCoord2D cartCoord2D::operator+(const cartCoord2D &rhs) const {
    cartCoord2D temp;
    temp.coords[0] = coords[0] + rhs.coords[0];
    temp.coords[1] = coords[1] + rhs.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D cartCoord2D::operator-(const cartCoord2D &rhs) const {
    cartCoord2D temp;
    temp.coords[0] = coords[0] - rhs.coords[0];
    temp.coords[1] = coords[1] - rhs.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D numDvd(const cartCoord2D &dividend, 
        const float divisor) {
    if(divisor == 0) {
        printf("divisor cannot be 0.\n");
        return cartCoord2D(0,0);
    } else {
        return cartCoord2D(dividend.coords[0]/divisor,dividend.coords[1]/divisor);
    }
}

__host__ __device__ cartCoord2D numMul(const float lambda, const cartCoord2D &pnt) {
    return cartCoord2D(lambda*pnt.coords[0],lambda*pnt.coords[1]);
}

__host__ __device__ float Psi_L(const cartCoord pnt) {
    return 1.0/(4*PI*pnt.nrm2());
}

__host__ __device__ float N_1(const cartCoord2D pnt) {
    return pnt.coords[0];
}

__host__ __device__ float N_2(const cartCoord2D pnt) {
    return pnt.coords[1];
}

__host__ __device__ float N_3(const cartCoord2D pnt) {
    return 1-pnt.coords[0]-pnt.coords[1];
}

__host__ __device__ float pN1pXi1(const cartCoord2D pnt) {
    return 1.0;
}

__host__ __device__ float pN1pXi2(const cartCoord2D pnt) {
    return 0;
}

__host__ __device__ float pN2pXi1(const cartCoord2D pnt) {
    return 0;
}

__host__ __device__ float pN2pXi2(const cartCoord2D pnt) {
    return 1.0;
}

__host__ __device__ float pN3pXi1(const cartCoord2D pnt) {
    return -1.0;
}

__host__ __device__ float pN3pXi2(const cartCoord2D pnt) {
    return -1.0;
}

__host__ __device__ cartCoord xiToElem(const cartCoord pnt1, const cartCoord pnt2,
        const cartCoord pnt3, const cartCoord2D localPnt) {
    return numMul(N_1(localPnt),pnt1)+numMul(N_2(localPnt),pnt2)
            +numMul(N_3(localPnt),pnt3); 
}

__host__ __device__ cartCoord pRvpXi1TimespRvpXi2(const cartCoord pnt1, const cartCoord pnt2, 
        const cartCoord pnt3) {
    return (pnt1-pnt3)*(pnt2-pnt3);
}

__host__ __device__ cartCoord2D etaToRhoTheta(const cartCoord2D s) {
    cartCoord2D temp;
    temp.coords[0] = 0.5+0.5*s.coords[0];
    temp.coords[1] = 0.5+0.5*s.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D rhoThetaToXi_3(const cartCoord2D s) {
    cartCoord2D temp;
    temp.coords[0] = s.coords[0]*(1-s.coords[1]); //rho*(1-theta)
    temp.coords[1] = s.coords[0]*s.coords[1];
    return temp;
}

__host__ __device__ cartCoord2D rhoThetaToXi_1(const cartCoord2D s) {
    cartCoord2D temp;
    temp.coords[0] = 1-s.coords[0];
    temp.coords[1] = s.coords[0]*(1-s.coords[1]);
    return temp;
}

__host__ __device__ cartCoord2D rhoThetaToXi_2(const cartCoord2D s) {
    cartCoord2D  temp;
    temp.coords[0] = s.coords[0]*s.coords[1];
    temp.coords[1] = 1-s.coords[0];
    return temp;
}

