#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "numerical.h"
#include "mesh.h"

ostream& operator<<(ostream &out, const hipFloatComplex &rhs) {
    out << "(" << hipCrealf(rhs) << "," << hipCimagf(rhs) << ")";
    return out;
}

__host__ __device__ void printComplexMatrix(hipFloatComplex *A, const int row, const int col, 
        const int lda) {
	float x, y;
	int i, j;
	for (i = 0;i < row;i++) {
		for (j = 0;j < col;j++) {
			x = hipCrealf(A[IDXC0(i, j, lda)]);
			y = hipCimagf(A[IDXC0(i, j, lda)]);
			printf("(%f,%f) ", x, y);
		}
		printf("\n");
	}		
}


//Gaussian cartCoords generation
gaussQuad::gaussQuad(const gaussQuad &rhs) {
    n = rhs.n;
    if(evalPnts != NULL) {
        delete[] evalPnts;
    }
    evalPnts = new float[n];
    for(int i=0;i<n;i++) {
        evalPnts[i] = rhs.evalPnts[i];
    }
    if(wgts != NULL) {
        delete[] evalPnts;
    }
    wgts = new float[n];
    for(int i=0;i<n;i++) {
        wgts[i] = rhs.wgts[i];
    }
}

gaussQuad::gaussQuad(const int order): n(order) {
    evalPnts=new float[n];
    wgts=new float[n];
    genGaussParams();
}

gaussQuad::~gaussQuad() {
    if(evalPnts != NULL) {
        delete[] evalPnts;
    }
    if(wgts != NULL) {
        delete[] wgts;
    }
}

gaussQuad& gaussQuad::operator=(const gaussQuad &rhs) {
    n = rhs.n;
    if(evalPnts != NULL) {
        delete[] evalPnts;
    }
    evalPnts = new float[n];
    for(int i=0;i<n;i++) {
        evalPnts[i] = rhs.evalPnts[i];
    }
    if(wgts != NULL) {
        delete[] evalPnts;
    }
    wgts = new float[n];
    for(int i=0;i<n;i++) {
        wgts[i] = rhs.wgts[i];
    }
    return *this;
}

int gaussQuad::genGaussParams() {
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        evalPnts[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgts[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}

ostream& operator<<(ostream &out, const gaussQuad &rhs) {
    out << "Points: " << endl;
    for(int i=0;i<rhs.n;i++) {
        out << rhs.evalPnts[i] << " ";
    }
    out << endl;
    out << "Weights: " << endl;
    for(int i=0;i<rhs.n;i++) {
        out << rhs.wgts[i] << " ";
    }
    return out;
}