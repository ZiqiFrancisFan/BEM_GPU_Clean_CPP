#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "numerical.h"
#include "mesh.h"

//air density and speed of sound
__constant__ float rho = 1.2041;

__constant__ float c = 343.21;

//Integral points and weights
__constant__ float INTPNTS[INTORDER]; 

__constant__ float INTWGTS[INTORDER];

__global__ void test(float *init) {
    printFloatMatrix(INTPNTS,1,INTORDER,1);
    printFloatMatrix(INTWGTS,1,INTORDER,1);
    printf("rho: %f\n",rho);
    printf("c: %f\n",c);
}

int Test() {
    gaussQuad g;
    g.sendToDevice();
    float *a;
    CUDA_CALL(hipMalloc(&a,sizeof(float)));
    test<<<1,1>>>(a);
    CUDA_CALL(hipFree(a));
    return EXIT_SUCCESS;
}

ostream& operator<<(ostream &out, const hipFloatComplex &rhs) {
    out << "(" << hipCrealf(rhs) << "," << hipCimagf(rhs) << ")";
    return out;
}

__host__ __device__ hipFloatComplex angExpf(const float theta) {
    return make_hipFloatComplex(cosf(theta),sinf(theta));
}

__host__ __device__ hipFloatComplex expfc(const hipFloatComplex z) {
    hipFloatComplex ans;
    float zr = hipCrealf(z), zi = hipCimagf(z);
    ans = make_hipFloatComplex(exp(zr)*cosf(zi),exp(zr)*sinf(zi));
    return ans;
}

__host__ __device__ hipFloatComplex green(const float k, const float r) {
    float y = 4*PI*r;
    hipFloatComplex x = angExpf(-k*r);
    return make_hipFloatComplex(hipCrealf(x)/y,hipCimagf(x)/y);
}


__host__ __device__ void printComplexMatrix(hipFloatComplex *A, const int row, const int col, 
        const int lda) {
    float x, y;
    int i, j;
    for (i = 0;i < row;i++) {
        for (j = 0;j < col;j++) {
                x = hipCrealf(A[IDXC0(i,j,lda)]);
                y = hipCimagf(A[IDXC0(i,j,lda)]);
                printf("(%f,%f) ",x,y);
        }
        printf("\n");
    }		
}

__host__ __device__ void printFloatMatrix(float *A, const int row, const int col, const int lda) {
    int i, j;
    for (i = 0;i < row;i++) {
        for (j = 0;j < col;j++) {
            printf("%f ",A[IDXC0(i,j,lda)]);
        }
        printf("\n");
    }	
}


//Gaussian cartCoords generation
gaussQuad::gaussQuad(const gaussQuad &rhs) {
    n = rhs.n;
    if(evalPnts != NULL) {
        delete[] evalPnts;
    }
    evalPnts = new float[n];
    for(int i=0;i<n;i++) {
        evalPnts[i] = rhs.evalPnts[i];
    }
    if(wgts != NULL) {
        delete[] evalPnts;
    }
    wgts = new float[n];
    for(int i=0;i<n;i++) {
        wgts[i] = rhs.wgts[i];
    }
}

gaussQuad::gaussQuad(const int order): n(order) {
    evalPnts=new float[n];
    wgts=new float[n];
    genGaussParams();
}

gaussQuad::~gaussQuad() {
    if(evalPnts != NULL) {
        delete[] evalPnts;
    }
    if(wgts != NULL) {
        delete[] wgts;
    }
}

gaussQuad& gaussQuad::operator=(const gaussQuad &rhs) {
    n = rhs.n;
    if(evalPnts != NULL) {
        delete[] evalPnts;
    }
    evalPnts = new float[n];
    for(int i=0;i<n;i++) {
        evalPnts[i] = rhs.evalPnts[i];
    }
    if(wgts != NULL) {
        delete[] evalPnts;
    }
    wgts = new float[n];
    for(int i=0;i<n;i++) {
        wgts[i] = rhs.wgts[i];
    }
    return *this;
}

int gaussQuad::genGaussParams() {
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        evalPnts[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgts[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}

int gaussQuad::sendToDevice() {
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTPNTS),evalPnts,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTWGTS),wgts,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

ostream& operator<<(ostream &out, const gaussQuad &rhs) {
    out << "Points: " << endl;
    for(int i=0;i<rhs.n;i++) {
        out << rhs.evalPnts[i] << " ";
    }
    out << endl;
    out << "Weights: " << endl;
    for(int i=0;i<rhs.n;i++) {
        out << rhs.wgts[i] << " ";
    }
    return out;
}

