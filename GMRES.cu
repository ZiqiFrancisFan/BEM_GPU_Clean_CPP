#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include "GMRES.h"

extern "C" HOST void printMatrix(hipFloatComplex *A, const int row, const int col, 
        const int lda)
{
	float x, y;
	int i, j;
	for (i=0;i<row;i++) {
            for (j=0;j<col;j++) {
                x = hipCrealf(A[IDXC0(i,j,lda)]);
                y = hipCimagf(A[IDXC0(i,j,lda)]);
                printf("(%f,%f) ",x,y);
            }
            printf("\n");
	}
		
}

extern "C" HOST void Rsolver(const hipFloatComplex *R, const hipFloatComplex *b, 
        const int m, hipFloatComplex *x) 
{
    int i, j;
    hipFloatComplex temp[6];
    for(i=m-1;i>=0;i--) {
        temp[0] = b[i];
        temp[1] = R[IDXC0(i,i,m)];
        for(j=m-1;j>i;j--) {
            temp[2] = x[j];
            temp[3] = R[IDXC0(i,j,m)];
            temp[4] = hipCmulf(temp[2],temp[3]);
            temp[0] = hipCsubf(temp[0],temp[4]);
        }
        x[i] = hipCdivf(temp[0],temp[1]);
    }
}

extern "C" HOST void givens_coeffs(const hipFloatComplex rho, 
        const hipFloatComplex sigma, hipFloatComplex *c, hipFloatComplex *s) 
{
    hipFloatComplex rho_b = hipConjf(rho), sigma_b = hipConjf(sigma);
    float x, y, mag;
    mag = sqrt(pow(hipCabsf(rho),2)+pow(hipCabsf(sigma),2));
    x = hipCrealf(rho_b)/mag;
    y = hipCimagf(rho_b)/mag;
    *c = make_hipFloatComplex(x,y);
    x = hipCrealf(sigma_b)/mag;
    y = hipCimagf(sigma_b)/mag;
    *s = make_hipFloatComplex(x,y);
}

extern "C" HOST void apply_givens(const int m, const int k, hipFloatComplex *c, 
        hipFloatComplex *s, hipFloatComplex *h) 
{
    hipFloatComplex c_k, s_k, c_b, s_b, temp[7];
    float x, y;
    int i;
    for(i=0;i<k-1;i++) {
        temp[0] = hipCmulf(c[i],h[i]);
        temp[1] = hipCmulf(s[i],h[i+1]);
        temp[2] = hipCaddf(temp[0],temp[1]);
        c_b = hipConjf(c[i]);
        s_b = hipConjf(s[i]);
        x = hipCrealf(s_b);
        y = hipCimagf(s_b);
        temp[3] = make_hipFloatComplex(-x,-y);
        temp[4] = hipCmulf(temp[3],h[i]);
        temp[5] = hipCmulf(c_b,h[i+1]);
        temp[6] = hipCaddf(temp[4],temp[5]);
        h[i+1] = temp[6];
        h[i] = temp[2];
    }
    if(k<m) {
        givens_coeffs(h[k-1],h[k],&c_k,&s_k);
        c[k-1] = c_k;
        s[k-1] = s_k;
        temp[0] = hipCmulf(c_k,h[k-1]);
        temp[1] = hipCmulf(s_k,h[k]);
        h[k-1] = hipCaddf(temp[0],temp[1]);
        h[k] = make_hipFloatComplex(0,0);
    }
}

extern "C" HOST int arnoldi(const hipFloatComplex *A_h, const int k, const int m, 
        hipFloatComplex *Q_h, hipFloatComplex *H_h) 
{
    if(k>m) {
        printf("Error with input k.\n");
        return EXIT_FAILURE;
    }
    float v_real, v_imag;
    int i;
    float nrm;
    hipFloatComplex alpha, beta, prod;
    alpha = make_hipFloatComplex(1,0);
    beta = make_hipFloatComplex(0,0);
    hipFloatComplex *A_d, *q_d, *y_d;
    hipblasHandle_t handle;
    CUBLAS_CALL(hipblasCreate(&handle));
    CUDA_CALL(hipMalloc(&A_d,m*m*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A_h,m*m*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&q_d,m*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(q_d,&Q_h[IDXC0(0,k-1,m)],m*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&y_d,m*sizeof(hipFloatComplex)));
    CUBLAS_CALL(hipblasCgemv(handle,HIPBLAS_OP_N,m,m,&alpha,A_d,m,q_d,1,&beta,
            y_d,1)); //Aq
    CUDA_CALL(hipFree(A_d));
    for(i=1;i<=k;i++) {
        CUDA_CALL(hipMemcpy(q_d,&Q_h[IDXC0(0,i-1,m)],m*sizeof(hipFloatComplex),
                hipMemcpyHostToDevice));
        CUBLAS_CALL(hipblasCdotc(handle,m,q_d,1,y_d,1,&prod));
        H_h[IDXC0(i-1,k-1,m)] = prod;
        v_real = hipCrealf(prod);
        v_imag = hipCimagf(prod);
        alpha = make_hipFloatComplex(-v_real,-v_imag);
        CUBLAS_CALL(hipblasCaxpy(handle,m,&alpha,q_d,1,y_d,1));
    }
    CUBLAS_CALL(hipblasScnrm2(handle,m,y_d,1,&nrm));
    if(k<m) {
        H_h[IDXC0(k,k-1,m)] = make_hipFloatComplex(nrm,0);
        alpha = make_hipFloatComplex(1.0/nrm,0);
        CUDA_CALL(hipMemset(q_d,0,m*sizeof(hipFloatComplex)));
        CUBLAS_CALL(hipblasCaxpy(handle,m,&alpha,y_d,1,q_d,1));
        CUDA_CALL(hipMemcpy(&Q_h[IDXC0(0,k,m)],q_d,m*sizeof(hipFloatComplex),
                hipMemcpyDeviceToHost));
    }
    //printf("Q in arnoldi: \n");
    //printMatrix(Q_h,m,m,m);
    
    CUDA_CALL(hipFree(y_d));
    CUDA_CALL(hipFree(q_d));
    CUBLAS_CALL(hipblasDestroy(handle));
    
    return EXIT_SUCCESS;
}

int GMRES(const hipFloatComplex *A_h, const hipFloatComplex *b_h, const int m, 
        const int maxIter, const float threshold, hipFloatComplex *x_h)
{
    //printf("input x: \n");
    //printMatrix(x_h,m,1,m);
    int i, j, t;
    float x, y;
    hipblasHandle_t handle;
    hipFloatComplex alpha, beta;
    hipFloatComplex *Q_h = (hipFloatComplex*)malloc(m*m*sizeof(hipFloatComplex));
    hipFloatComplex *H_h = (hipFloatComplex*)malloc(m*m*sizeof(hipFloatComplex));
    hipFloatComplex *c = (hipFloatComplex*)malloc(m*sizeof(hipFloatComplex));
    hipFloatComplex *s = (hipFloatComplex*)malloc(m*sizeof(hipFloatComplex));
    hipFloatComplex *h = (hipFloatComplex*)malloc(m*sizeof(hipFloatComplex));
    hipFloatComplex *err_h = (hipFloatComplex*)malloc(m*sizeof(hipFloatComplex));
    hipFloatComplex *A_d, *x_d, *r_d, *q_d;
    CUDA_CALL(hipMalloc(&A_d,m*m*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A_h,m*m*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&x_d,m*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(x_d,x_h,m*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&r_d,m*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(r_d,b_h,m*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&q_d,m*sizeof(hipFloatComplex)));
    float nrm_b, nrm_r;
    CUBLAS_CALL(hipblasCreate(&handle));
    CUBLAS_CALL(hipblasScnrm2(handle,m,r_d,1,&nrm_b)); //norm of b vector
    alpha = make_hipFloatComplex(-1,0);
    beta = make_hipFloatComplex(1,0);
    CUBLAS_CALL(hipblasCgemv(handle,HIPBLAS_OP_N,m,m,&alpha,A_d,m,x_d,1,&beta,
            r_d,1)); //r = b-Ax
    CUDA_CALL(hipFree(A_d));
    CUBLAS_CALL(hipblasScnrm2(handle,m,r_d,1,&nrm_r)); //norm of r vector
    err_h[0] = make_hipFloatComplex(nrm_r,0);
    //printf("nrm_r=%f\n",nrm_r);
    if(nrm_r/nrm_b<threshold) {
        printf("The initial x is accurate enough.\n");
        CUDA_CALL(hipFree(q_d));
        CUDA_CALL(hipFree(r_d));
        CUDA_CALL(hipFree(x_d));
        free(err_h);
        free(h);
        free(s);
        free(c);
        free(H_h);
        free(Q_h);
        return EXIT_SUCCESS;
    }
    CUDA_CALL(hipMemset(q_d,0,m*sizeof(hipFloatComplex)));
    alpha = make_hipFloatComplex(1.0/nrm_r,0);
    CUBLAS_CALL(hipblasCaxpy(handle,m,&alpha,r_d,1,q_d,1));
    CUDA_CALL(hipMemcpy(&Q_h[IDXC0(0,0,m)],q_d,m*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    i = 1;
    while(1) {
        arnoldi(A_h,i,m,Q_h,H_h);
        //printf("H_h before givens: \n");
        //printMatrix(H_h,m,i,m);
        if(i<m) {
            CUDA_CALL(hipMemcpy(h,&H_h[IDXC0(0,i-1,m)],
                    (i+1)*sizeof(hipFloatComplex),hipMemcpyHostToHost));
            apply_givens(m,i,c,s,h);
            CUDA_CALL(hipMemcpy(&H_h[IDXC0(0,i-1,m)],h,
                    (i+1)*sizeof(hipFloatComplex),hipMemcpyHostToHost));
            alpha = hipConjf(s[i-1]);
            x = hipCrealf(alpha);
            y = hipCimagf(alpha);
            beta = make_hipFloatComplex(-x,-y);
            err_h[i] = hipCmulf(beta,err_h[i-1]);
            err_h[i-1] = hipCmulf(c[i-1],err_h[i-1]);
        } else {
            CUDA_CALL(hipMemcpy(h,&H_h[IDXC0(0,i-1,m)],m*sizeof(hipFloatComplex),
                    hipMemcpyHostToHost));
            apply_givens(m,i,c,s,h);
            CUDA_CALL(hipMemcpy(&H_h[IDXC0(0,i-1,m)],h,m*sizeof(hipFloatComplex),
                    hipMemcpyHostToHost));
        }
        if((i<m && hipCabsf(err_h[i])/nrm_b<threshold) || i>=maxIter) {
            break;
        }
        i++;
        printf("Iteration: %d\n",i);
    }
    //printf("c: \n");
    //printMatrix(c,m-1,1,m);
    //printf("s: \n");
    //printMatrix(s,m-1,1,m);
    //printf("err_h: \n");
    //printMatrix(err_h,m,1,m);
    t = i;
    hipFloatComplex *y_h = (hipFloatComplex*)malloc(t*sizeof(hipFloatComplex));
    hipFloatComplex *R_h = (hipFloatComplex*)malloc(t*t*sizeof(hipFloatComplex));
    for(i=0;i<t;i++) {
        for(j=0;j<t;j++) {
            R_h[IDXC0(i,j,t)] = H_h[IDXC0(i,j,m)];
        }
    }
    //printf("H_h: \n");
    //printMatrix(H_h,m,m,m);
    //printf("R_h: \n");
    //printMatrix(R_h,i,i,i);
    Rsolver(R_h,err_h,t,y_h);
    free(R_h);
    //printf("y_h: \n");
    //printMatrix(y_h,t,1,t);
    
    hipFloatComplex *y_d, *Q_d;
    CUDA_CALL(hipMalloc(&y_d,t*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(y_d,y_h,t*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&Q_d,m*t*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(Q_d,Q_h,m*t*sizeof(hipFloatComplex),
            hipMemcpyHostToDevice));
    alpha = make_hipFloatComplex(1,0);
    beta = make_hipFloatComplex(1,0);
    CUBLAS_CALL(hipblasCgemv(handle,HIPBLAS_OP_N,m,t,&alpha,Q_d,m,y_d,1,&beta,
            x_d,1));
    CUDA_CALL(hipMemcpy(x_h,x_d,m*sizeof(hipFloatComplex),
            hipMemcpyDeviceToHost));
    CUBLAS_CALL(hipblasDestroy(handle));
    CUDA_CALL(hipFree(y_d));
    CUDA_CALL(hipFree(Q_d));
    CUDA_CALL(hipFree(q_d));
    CUDA_CALL(hipFree(r_d));
    CUDA_CALL(hipFree(x_d));
    free(err_h);
    free(h);
    free(s);
    free(c);
    free(H_h);
    free(Q_h);
    return EXIT_SUCCESS;
}


